#include "hip/hip_runtime.h"
// Ensure printing of CUDA runtime errors to console
#define HIPCUB_STDERR

#include <iostream>
#include <stdio.h>
#include <hiprand.h>

#include <hip/hip_runtime.h>
#include <cub/util_allocator.cuh>
#include "cub/test/test_util.h"

#include "crystal.cuh"

#include "utils/generator.h"
#include "utils/gpu_utils.h"

using namespace std;

#define DEBUG 1

template<int BLOCK_THREADS, int ITEMS_PER_THREAD>
__global__ void build_kernel(int *dim_key, int *dim_val, int num_tuples, int *hash_table, int num_slots) {
  int items[ITEMS_PER_THREAD];
  int items2[ITEMS_PER_THREAD];
  int selection_flags[ITEMS_PER_THREAD];

  int tile_offset = blockIdx.x * TILE_SIZE;
  int num_tiles = (num_tuples + TILE_SIZE - 1) / TILE_SIZE;
  int num_tile_items = TILE_SIZE;

  if (blockIdx.x == num_tiles - 1) {
    num_tile_items = num_tuples - tile_offset;
  }

  InitFlags<BLOCK_THREADS, ITEMS_PER_THREAD>(selection_flags);
  BlockLoad<int, BLOCK_THREADS, ITEMS_PER_THREAD>(dim_key + tile_offset, items, num_tile_items);
  BlockLoad<int, BLOCK_THREADS, ITEMS_PER_THREAD>(dim_val + tile_offset, items2, num_tile_items);
  BlockBuildSelectivePHT_2<int, int, BLOCK_THREADS, ITEMS_PER_THREAD>(items, items2, selection_flags, 
      hash_table, num_slots, num_tile_items);
}

template<int BLOCK_THREADS, int ITEMS_PER_THREAD>
__global__ void probe_kernel(int *fact_fkey, int *fact_val, int num_tuples, 
    int *hash_table, int num_slots, unsigned long long *res) {
  // Load a tile striped across threads
  int selection_flags[ITEMS_PER_THREAD];
  int keys[ITEMS_PER_THREAD];
  int vals[ITEMS_PER_THREAD];
  int join_vals[ITEMS_PER_THREAD];

  unsigned long long sum = 0;

  int tile_offset = blockIdx.x * TILE_SIZE;
  int num_tiles = (num_tuples+ TILE_SIZE - 1) / TILE_SIZE;
  int num_tile_items = TILE_SIZE;

  if (blockIdx.x == num_tiles - 1) {
    num_tile_items = num_tuples - tile_offset;
  }

  InitFlags<BLOCK_THREADS, ITEMS_PER_THREAD>(selection_flags);
  BlockLoad<int, BLOCK_THREADS, ITEMS_PER_THREAD>(fact_fkey + tile_offset, keys, num_tile_items);
  BlockLoad<int, BLOCK_THREADS, ITEMS_PER_THREAD>(fact_val + tile_offset, vals, num_tile_items);

  BlockProbeAndPHT_2<int, int, BLOCK_THREADS, ITEMS_PER_THREAD>(keys, join_vals, selection_flags,
      hash_table, num_slots, num_tile_items);

  #pragma unroll
  for (int ITEM = 0; ITEM < ITEMS_PER_THREAD; ++ITEM)
  {
    if ((threadIdx.x + (BLOCK_THREADS * ITEM) < num_tile_items))
      if (selection_flags[ITEM])
        sum += vals[ITEM] * join_vals[ITEM];
  }

  __syncthreads();

  static __shared__ long long buffer[32];
  unsigned long long aggregate = BlockSum<long long, BLOCK_THREADS, ITEMS_PER_THREAD>(sum, (long long*)buffer);
  __syncthreads();

  if (threadIdx.x == 0) {
    atomicAdd(res, aggregate);
  }
}

struct TimeKeeper {
  float time_build;
  float time_probe;
  float time_extra;
  float time_total;
};

TimeKeeper hashJoin(int* d_dim_key, int* d_dim_val, int* d_fact_fkey, int* d_fact_val, int num_dim, int num_fact, hipcub::CachingDeviceAllocator&  g_allocator) {
  SETUP_TIMING();

  int* hash_table = NULL;
  unsigned long long* res;
  int num_slots = num_dim;
  float time_build, time_probe, time_memset, time_memset2;

  ALLOCATE(hash_table, sizeof(int) * 2 * num_dim);
  ALLOCATE(res, sizeof(long long));

  TIME_FUNC(hipMemset(hash_table, 0, num_slots * sizeof(int) * 2), time_memset);
  TIME_FUNC(hipMemset(res, 0, sizeof(long long)), time_memset2);

  int tile_items = 128*4;

  TIME_FUNC((build_kernel<128, 4><<<(num_dim + tile_items - 1)/tile_items, 128>>>(d_dim_key, d_dim_val, num_dim, hash_table, num_slots)), time_build);
  TIME_FUNC((probe_kernel<128, 4><<<(num_fact + tile_items - 1)/tile_items, 128>>>(d_fact_fkey, d_fact_val, num_fact, hash_table, num_slots, res)), time_probe);

#if DEBUG
  cout << "{" << "\"time_memset\":" << time_memset
      << ",\"time_build\"" << time_build
      << ",\"time_probe\":" << time_probe << "}" << endl;
#endif

  CLEANUP(hash_table);
  CLEANUP(res);

  TimeKeeper t = {time_build, time_probe, time_memset, time_build + time_probe + time_memset};
  return t;
}

//---------------------------------------------------------------------
// Globals, constants and typedefs
//---------------------------------------------------------------------

bool                    g_verbose = false;  // Whether to display input/output to console
hipcub::CachingDeviceAllocator  g_allocator(true);  // Caching allocator for device memory


#define CLEANUP(vec) if(vec)CubDebugExit(g_allocator.DeviceFree(vec))

//---------------------------------------------------------------------
// Main
//---------------------------------------------------------------------
int main(int argc, char** argv)
{
  int num_fact           = 256 * 1<<20;
  int num_dim            = 16 * 1<<20;
  int num_trials         = 3;

  // Initialize command line
  CommandLineArgs args(argc, argv);
  args.GetCmdLineArgument("n", num_fact);
  args.GetCmdLineArgument("d", num_dim);
  args.GetCmdLineArgument("t", num_trials);

  // Print usage
  if (args.CheckCmdLineFlag("help"))
  {
    printf("%s "
        "[--n=<num fact>] "
        "[--d=<num dim>] "
        "[--t=<num trials>] "
        "[--device=<device-id>] "
        "[--v] "
        "\n", argv[0]);
    exit(0);
  }

  int log2 = 0;
  int num_dim_dup = num_dim >> 1;
  while (num_dim_dup) {
    num_dim_dup >>= 1;
    log2 += 1;
  }

  // Initialize device
  CubDebugExit(args.DeviceInit());

  // Allocate problem device arrays
  int *d_dim_key = NULL;
  int *d_dim_val = NULL;
  int *d_fact_fkey = NULL;
  int *d_fact_val = NULL;
  CubDebugExit(g_allocator.DeviceAllocate((void**)&d_dim_key, sizeof(int) * num_dim));
  CubDebugExit(g_allocator.DeviceAllocate((void**)&d_dim_val, sizeof(int) * num_dim));
  CubDebugExit(g_allocator.DeviceAllocate((void**)&d_fact_fkey, sizeof(int) * num_fact));
  CubDebugExit(g_allocator.DeviceAllocate((void**)&d_fact_val, sizeof(int) * num_fact));

  int *h_dim_key = NULL;
  int *h_dim_val = NULL;
  int *h_fact_fkey = NULL;
  int *h_fact_val = NULL;

  create_relation_pk(h_dim_key, h_dim_val, num_dim);
  create_relation_fk(h_fact_fkey, h_fact_val, num_fact, num_dim);

  CubDebugExit(hipMemcpy(d_dim_key, h_dim_key, sizeof(int) * num_dim, hipMemcpyHostToDevice));
  CubDebugExit(hipMemcpy(d_dim_val, h_dim_val, sizeof(int) * num_dim, hipMemcpyHostToDevice));
  CubDebugExit(hipMemcpy(d_fact_fkey, h_fact_fkey, sizeof(int) * num_fact, hipMemcpyHostToDevice));
  CubDebugExit(hipMemcpy(d_fact_val, h_fact_val, sizeof(int) * num_fact, hipMemcpyHostToDevice));

  for (int j = 0; j < num_trials; j++) {
    TimeKeeper t = hashJoin(d_dim_key, d_dim_val, d_fact_fkey, d_fact_val, num_dim, num_fact, g_allocator);
    cout<< "{"
        << "\"num_dim\":" << num_dim
        << ",\"num_fact\":" << num_fact
        << ",\"radix\":" << 0
        << ",\"time_partition_build\":" << 0
        << ",\"time_partition_probe\":" << 0
        << ",\"time_partition_total\":" << 0
        << ",\"time_build\":" << t.time_build
        << ",\"time_probe\":" << t.time_probe
        << ",\"time_extra\":" << t.time_extra
        << ",\"time_join_total\":" << t.time_total
        << "}" << endl;
  }

  CLEANUP(d_dim_key);
  CLEANUP(d_dim_val);
  CLEANUP(d_fact_fkey);
  CLEANUP(d_fact_val);

  return 0;
}

