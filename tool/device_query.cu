#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>
#include <stdio.h>

#define HANDLER_ERROR_MSG(msg) (hipError_t(msg, __FILE__, __LINE__))

void hipError_t(const char* msg, const char* file, int line) {
	hipError_t err = hipGetLastError();
	if (hipSuccess != err) {
		fprintf(stderr, "%s: %s in %s at line %d\n", msg, hipGetErrorString(err), file, line);
		exit(EXIT_FAILURE);
	}
}

void deviceProperties() {
	hipDeviceProp_t prop;

	int count, driverVersion = 0, runtimeVersion = 0;
	hipGetDeviceCount(&count);
	HANDLER_ERROR_MSG("device count");
	for (int i = 0; i < count; i++) {
		hipGetDeviceProperties(&prop, i);
		HANDLER_ERROR_MSG("device prop");

		printf("   --- General Information for device %d ---\n", i);
		printf("Name:  %s\n", prop.name);
		hipDriverGetVersion(&driverVersion);
		hipRuntimeGetVersion(&runtimeVersion);
		printf("CUDA Driver Version  %d.%d\n", driverVersion / 1000, (driverVersion % 100) / 10);
		printf("Runtime Version %d.%d\n", runtimeVersion / 1000, (runtimeVersion % 100) / 10);

		// The compute capability of the device
		printf("Compute capability:  %d.%d\n", prop.major, prop.minor);
		// The clock frequency, how fast the actual processors in the GPU are
		// going
		printf("\n");
		printf("Clock rate:  %.0f MHz (%.0f GHz)\n", prop.clockRate * 1e-3f, prop.clockRate * 1e-6f);
		printf("\n");
		// The device can concurrently copy memory and execute a kernel.

		printf("Concurrent kernels:  %s \n", prop.concurrentKernels ? "Enabled" : "Disabled");
#if CUDART_VERSION >= 5000
		printf("Concurrent copy and kernel execution %s with %d copy engine(s)\n",
		       (prop.deviceOverlap ? "Enabled" : "Disabled"),
		       prop.asyncEngineCount);
#endif
		// Specified whether there is a run time limit on kernels
		printf("Kernel execution timeout :  %s \n", prop.kernelExecTimeoutEnabled ? "Enabled" : "Disabled");
		// The device can use mapped memory
		printf("Integrated GPU sharing Host Memory: %s\n", prop.integrated ? "Enabled" : "Disabled");
		printf("Support host page-locked memory mapping: %s\n", prop.canMapHostMemory ? "Enabled" : "NoDisabled");

		printf("\n   --- Memory Information for device %d ---\n", i);

#if CUDART_VERSION >= 5000
		//  how fast the memory in the GPU is operating
		printf("Memory Clock rate: %f Ghz\n", prop.memoryClockRate * 10e-7);
		// how many bits of memory are actually being tranferred for each
		// memory clock cycle
		printf("Memory Bus Width:  %d-bit\n", prop.memoryBusWidth);

#endif

		printf("Total global mem:  %lf Mbytes (%ld bytes) \n", prop.totalGlobalMem / 1048576.0, prop.totalGlobalMem);
		printf("Total constant Mem:  %ld bytes\n", prop.totalConstMem);
		printf("Max mem pitch:  %ld bytes\n", prop.memPitch);

		printf("\n   --- MP Information for device %d ---\n", i);
		printf("Multiprocessor count:  %d\n", prop.multiProcessorCount);
		printf("Shared mem per block:  %ld bytes \n", prop.sharedMemPerBlock);
		printf("Registers per block:  %d\n", prop.regsPerBlock);
		printf("Threads in warp:  %d\n", prop.warpSize);
#if CUDART_VERSION >= 5000
		printf("Max threads per multiprocessor: %d\n", prop.maxThreadsPerMultiProcessor);
#endif
		printf("Max threads per block:  %d\n", prop.maxThreadsPerBlock);
		printf("Max thread dimensions:  (%d, %d, %d)\n",
		       prop.maxThreadsDim[0],
		       prop.maxThreadsDim[1],
		       prop.maxThreadsDim[2]);
		printf("Max grid dimensions:  (%d, %d, %d)\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
		printf("\n");
	}
}

int main(void) { deviceProperties(); }