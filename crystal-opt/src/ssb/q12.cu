#include "hip/hip_runtime.h"
// MIT License

// Copyright (c) 2023 Jiashen Cao

// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:

// The above copyright notice and this permission notice shall be included in all
// copies or substantial portions of the Software.

// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE.

// Ensure printing of CUDA runtime errors to console
#define HIPCUB_STDERR

#include <hiprand.h>
#include <iostream>
#include <stdio.h>

#include "cub/test/test_util.h"
#include <cub/util_allocator.cuh>
#include <hip/hip_runtime.h>

#include "crystal.cuh"

#include "gpu_utils.h"
#include "ssb_utils.h"

using namespace std;

/**
 * Globals, constants and typedefs
 */
bool g_verbose = false; // Whether to display input/output to console
hipcub::CachingDeviceAllocator
    g_allocator(true); // Caching allocator for device memory

template <int BLOCK_THREADS, int ITEMS_PER_THREAD>
__global__ void DeviceSelectIf(int *lo_orderdate, int *lo_discount,
                               int *lo_quantity, int *lo_extendedprice,
                               int lo_num_entries,
                               unsigned long long *revenue) {
  // Load a segment of consecutive items that are blocked across threads
  int items[ITEMS_PER_THREAD];
  int selection_flags[ITEMS_PER_THREAD];
  int items2[ITEMS_PER_THREAD];

  long long sum = 0;

  int tile_offset = blockIdx.x * TILE_SIZE;
  int num_tiles = (lo_num_entries + TILE_SIZE - 1) / TILE_SIZE;
  int num_tile_items = TILE_SIZE;

  if (blockIdx.x == num_tiles - 1) {
    num_tile_items = lo_num_entries - tile_offset;
  }

  BlockLoad<int, BLOCK_THREADS, ITEMS_PER_THREAD>(lo_orderdate + tile_offset,
                                                  items, num_tile_items);
  BlockPredGTE<int, BLOCK_THREADS, ITEMS_PER_THREAD>(
      items, 19940101, selection_flags, num_tile_items);
  BlockPredAndLTE<int, BLOCK_THREADS, ITEMS_PER_THREAD>(
      items, 19940131, selection_flags, num_tile_items);

  BlockPredLoad<int, BLOCK_THREADS, ITEMS_PER_THREAD>(
      lo_quantity + tile_offset, items, num_tile_items, selection_flags);
  BlockPredAndGTE<int, BLOCK_THREADS, ITEMS_PER_THREAD>(
      items, 26, selection_flags, num_tile_items);
  BlockPredAndLTE<int, BLOCK_THREADS, ITEMS_PER_THREAD>(
      items, 35, selection_flags, num_tile_items);

  BlockPredLoad<int, BLOCK_THREADS, ITEMS_PER_THREAD>(
      lo_discount + tile_offset, items, num_tile_items, selection_flags);
  BlockPredAndGTE<int, BLOCK_THREADS, ITEMS_PER_THREAD>(
      items, 4, selection_flags, num_tile_items);
  BlockPredAndLTE<int, BLOCK_THREADS, ITEMS_PER_THREAD>(
      items, 6, selection_flags, num_tile_items);

  BlockPredLoad<int, BLOCK_THREADS, ITEMS_PER_THREAD>(
      lo_extendedprice + tile_offset, items2, num_tile_items, selection_flags);

#pragma unroll
  for (int ITEM = 0; ITEM < ITEMS_PER_THREAD; ++ITEM) {
    if (threadIdx.x + (BLOCK_THREADS * ITEM) < num_tile_items)
      if (selection_flags[ITEM])
        sum += items[ITEM] * items2[ITEM];
  }

  __syncthreads();

  static __shared__ long long buffer[32];
  unsigned long long aggregate =
      BlockSum<long long, BLOCK_THREADS, ITEMS_PER_THREAD>(sum,
                                                           (long long *)buffer);
  __syncthreads();

  if (threadIdx.x == 0) {
    atomicAdd(revenue, aggregate);
  }
}

float runQuery(int *lo_orderdate, int *lo_discount, int *lo_quantity,
               int *lo_extendedprice, int lo_num_entries,
               hipcub::CachingDeviceAllocator &g_allocator) {
  SETUP_TIMING();

  float time_query;
  chrono::high_resolution_clock::time_point st, finish;
  st = chrono::high_resolution_clock::now();

  hipEventRecord(start, 0);

  unsigned long long *d_sum = NULL;
  CubDebugExit(g_allocator.DeviceAllocate((void **)&d_sum, sizeof(long long)));

  hipMemset(d_sum, 0, sizeof(long long));

  // Run
  int tile_items = 128 * 4;
  DeviceSelectIf<128, 4>
      <<<(lo_num_entries + tile_items - 1) / tile_items, 128>>>(
          lo_orderdate, lo_discount, lo_quantity, lo_extendedprice,
          lo_num_entries, d_sum);

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time_query, start, stop);

  unsigned long long revenue;
  CubDebugExit(
      hipMemcpy(&revenue, d_sum, sizeof(long long), hipMemcpyDeviceToHost));

  finish = chrono::high_resolution_clock::now();
  std::chrono::duration<double> diff = finish - st;

  cout << "Revenue: " << revenue << endl;
  cout << "Time Taken Total: " << diff.count() * 1000 << endl;

  CLEANUP(d_sum);

  return time_query;
}

/**
 * Main
 */
int main(int argc, char **argv) {
  int num_trials = 3;

  // Initialize command line
  CommandLineArgs args(argc, argv);
  args.GetCmdLineArgument("t", num_trials);

  // Print usage
  if (args.CheckCmdLineFlag("help")) {
    printf("%s "
           "[--t=<num trials>] "
           "[--v] "
           "\n",
           argv[0]);
    exit(0);
  }

  // Initialize device
  CubDebugExit(args.DeviceInit());

  int *h_lo_orderdate = loadColumn<int>("lo_orderdate", LO_LEN);
  int *h_lo_discount = loadColumn<int>("lo_discount", LO_LEN);
  int *h_lo_quantity = loadColumn<int>("lo_quantity", LO_LEN);
  int *h_lo_extendedprice = loadColumn<int>("lo_extendedprice", LO_LEN);
  int *h_d_datekey = loadColumn<int>("d_datekey", D_LEN);
  int *h_d_year = loadColumn<int>("d_year", D_LEN);

  cout << "** LOADED DATA **" << endl;

  int *d_lo_orderdate = loadToGPU<int>(h_lo_orderdate, LO_LEN, g_allocator);
  int *d_lo_discount = loadToGPU<int>(h_lo_discount, LO_LEN, g_allocator);
  int *d_lo_quantity = loadToGPU<int>(h_lo_quantity, LO_LEN, g_allocator);
  int *d_lo_extendedprice =
      loadToGPU<int>(h_lo_extendedprice, LO_LEN, g_allocator);
  int *d_d_datekey = loadToGPU<int>(h_d_datekey, D_LEN, g_allocator);
  int *d_d_year = loadToGPU<int>(h_d_year, D_LEN, g_allocator);

  cout << "** LOADED DATA TO GPU **" << endl;

  for (int t = 0; t < num_trials; t++) {
    float time_query;
    time_query = runQuery(d_lo_orderdate, d_lo_discount, d_lo_quantity,
                          d_lo_extendedprice, LO_LEN, g_allocator);
    cout << "{"
         << "\"query\":12"
         << ",\"time_query\":" << time_query << "}" << endl;
  }

  return 0;
}
