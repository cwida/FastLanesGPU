#include "hip/hip_runtime.h"
// MIT License

// Copyright (c) 2023 Jiashen Cao

// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:

// The above copyright notice and this permission notice shall be included in all
// copies or substantial portions of the Software.

// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE.

// Ensure printing of CUDA runtime errors to console
#define HIPCUB_STDERR

#include <iostream>
#include <stdio.h>
#include <hiprand.h>
#include <cmath>

#include <hip/hip_runtime.h>
#include <cub/util_allocator.cuh>
#include "cub/test/test_util.h"

#include "crystal.cuh"

#include "utils/gpu_utils.h"

using namespace std;


//---------------------------------------------------------------------
// Implements Projection Operator
// There are two variants: dot-product and sigmoid
//---------------------------------------------------------------------

bool                    g_verbose = false;  // Whether to display input/output to console
hipcub::CachingDeviceAllocator  g_allocator(true);  // Caching allocator for device memory

template<int BLOCK_THREADS, int ITEMS_PER_THREAD>
__global__ void project(float* in1, float* in2, float* out, int num_items)
{
  float items[ITEMS_PER_THREAD];
  float items2[ITEMS_PER_THREAD];
  float res[ITEMS_PER_THREAD];

  int tile_offset = blockIdx.x * TILE_SIZE;
  int num_tiles = (num_items + TILE_SIZE - 1) / TILE_SIZE;
  int num_tile_items = TILE_SIZE;

  if (blockIdx.x == num_tiles - 1) {
    num_tile_items = num_items - tile_offset;
  }

  BlockLoad<float, BLOCK_THREADS, ITEMS_PER_THREAD>(in1 + tile_offset, items, num_tile_items);
  BlockLoad<float, BLOCK_THREADS, ITEMS_PER_THREAD>(in2 + tile_offset, items2, num_tile_items);

  #pragma unroll
  for (int ITEM = 0; ITEM < ITEMS_PER_THREAD; ITEM++) {
    if (threadIdx.x + (ITEM * BLOCK_THREADS) < num_tile_items) {
      res[ITEM] = 2*items[ITEM] + 3*items2[ITEM];
    }
  }

  BlockStore<float, BLOCK_THREADS, ITEMS_PER_THREAD>(out + tile_offset, res, num_tile_items);
}

template<int BLOCK_THREADS, int ITEMS_PER_THREAD>
__global__ void projectSigmoid(float* in1, float* in2, float* out, int num_items)
{
  float items[ITEMS_PER_THREAD];
  float items2[ITEMS_PER_THREAD];
  float res[ITEMS_PER_THREAD];

  int tile_offset = blockIdx.x * TILE_SIZE;
  int num_tiles = (num_items + TILE_SIZE - 1) / TILE_SIZE;
  int num_tile_items = TILE_SIZE;

  if (blockIdx.x == num_tiles - 1) {
    num_tile_items = num_items - tile_offset;
  }

  BlockLoad<float, BLOCK_THREADS, ITEMS_PER_THREAD>(in1 + tile_offset, items, num_tile_items);
  BlockLoad<float, BLOCK_THREADS, ITEMS_PER_THREAD>(in2 + tile_offset, items2, num_tile_items);

  #pragma unroll
  for (int ITEM = 0; ITEM < ITEMS_PER_THREAD; ITEM++) {
    if (threadIdx.x + (ITEM * BLOCK_THREADS) < num_tile_items) {
      res[ITEM] = 1.0f / (1.0f + expf(-2*items[ITEM] -3*items2[ITEM]));
    }
  }

  BlockStore<float, BLOCK_THREADS, ITEMS_PER_THREAD>(out + tile_offset, res, num_tile_items);
}


float projectGPU(float* in1, float* in2, float* out, int num_items) {
  SETUP_TIMING();

  float time_proj;
  int tile_items = 128*4;
  int num_blocks = (num_items + tile_items - 1)/tile_items;
  TIME_FUNC((project<128,4><<<num_blocks, 128>>>(in1, in2, out, num_items)), time_proj);

  return time_proj;
}

float projectSigmoidGPU(float* in1, float* in2, float* out, int num_items) {
  SETUP_TIMING();

  float time_proj;
  int tile_items = 128*4;
  int num_blocks = (num_items + tile_items - 1)/tile_items;
  TIME_FUNC((projectSigmoid<128,4><<<num_blocks, 128>>>(in1, in2, out, num_items)), time_proj);

  return time_proj;
}

/**
 * Main
 */
int main(int argc, char** argv)
{
  int num_items           = 1<<28;
  int num_trials          = 1;

  // Initialize command line
  CommandLineArgs args(argc, argv);
  args.GetCmdLineArgument("n", num_items);
  args.GetCmdLineArgument("t", num_trials);

  // Print usage
  if (args.CheckCmdLineFlag("help"))
  {
      printf("%s "
          "[--n=<input items>] "
          "[--t=<num trials>] "
          "[--device=<device-id>] "
          "[--v] "
          "\n", argv[0]);
      exit(0);
  }

  // Initialize device
  CubDebugExit(args.DeviceInit());

  // Allocate problem device arrays
  float *d_in1 = NULL;
  CubDebugExit(g_allocator.DeviceAllocate((void**)&d_in1, sizeof(float) * num_items));

  float *d_in2 = NULL;
  CubDebugExit(g_allocator.DeviceAllocate((void**)&d_in2, sizeof(float) * num_items));

  float  *d_out = NULL;
  CubDebugExit(g_allocator.DeviceAllocate((void**)&d_out, sizeof(float) * num_items));

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hiprandGenerator_t generator;
  int seed = 0;
  hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_DEFAULT);
  hiprandSetPseudoRandomGeneratorSeed(generator,seed);
  hiprandGenerateUniform(generator, d_in1, num_items);
  hiprandGenerateUniform(generator, d_in2, num_items);

  float time_proj_gpu;
  float time_proj_sigmoid_gpu;  

  for (int t = 0; t < num_trials; t++) {
    time_proj_gpu = projectGPU(d_in1, d_in2, d_out, num_items);
    time_proj_sigmoid_gpu = projectSigmoidGPU(d_in1, d_in2, d_out, num_items);

    cout<< "{"
        << "\"time_proj_gpu\":" << time_proj_gpu
        << ",\"time_proj_sigmoid_gpu\":" << time_proj_sigmoid_gpu
        << "}" << endl;
  }

  // Cleanup
  if (d_in1) CubDebugExit(g_allocator.DeviceFree(d_in1));
  if (d_in2) CubDebugExit(g_allocator.DeviceFree(d_in2));
  if (d_out) CubDebugExit(g_allocator.DeviceFree(d_out));

  return 0;
}

