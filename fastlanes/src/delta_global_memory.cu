#include "hip/hip_runtime.h"
#include "crystal/crystal.cuh"
#include "cub/test/test_util.h"
#include "fls_gen/unpack/unpack.cuh"
#include "gpu_utils.h"
#include "ssb_utils.h"
#include <fls_gen/pack/pack.hpp>
#include <fls_gen/rsum/rsum.cuh>
#include <fls_gen/transpose/transpose.hpp>
#include <fls_gen/unrsum/unrsum.hpp>
#include <iostream>
#include <stdio.h>

using namespace std;
using namespace fastlanes;
using namespace fastlanes::gpu;

struct QueryMtd {
	n_t      n_vec;
	uint8_t  bw;
	n_t      n_tup;
	uint64_t result;
};

template <int BLOCK_THREADS, int ITEMS_PER_THREAD>
__global__ void
QueryKernel(const uint32_t* base_col, const uint32_t* bitpacked_col, QueryMtd query_mtd, uint32_t* out) {
	int TILE_SIZE = BLOCK_THREADS * ITEMS_PER_THREAD;
	// Load a segment of consecutive items that are blocked across threads
	// uint32_t items[ITEMS_PER_THREAD];

	static __shared__ uint32_t unpacked[1024];

	int tile_offset = blockIdx.x * TILE_SIZE;

	int bitpacked_col_tile_offset = blockIdx.x * query_mtd.bw * 32;
	unpack_device(bitpacked_col + bitpacked_col_tile_offset, unpacked, query_mtd.bw);

	int based_col_tile_offset = blockIdx.x * 32;
	d_rsum_32(unpacked, out + tile_offset, base_col + based_col_tile_offset);
}

template <int BLOCK_THREADS, int ITEMS_PER_THREAD>
float decode(const uint32_t*              org_col,
             uint32_t*                    decoded_arr,
             const uint32_t*              base_col,
             const uint32_t*              bitpacked_col,
             QueryMtd                     hardcoded,
             hipcub::CachingDeviceAllocator& g_allocator) {
	// int TILE_SIZE = BLOCK_THREADS * ITEMS_PER_THREAD;

	SETUP_TIMING();
	float                                     time_query;
	chrono::high_resolution_clock::time_point st, finish;
	st                 = chrono::high_resolution_clock::now();
	uint32_t* d_result = NULL;
	CubDebugExit(g_allocator.DeviceAllocate((void**)&d_result, 1024 * 1024 * 1024));
	CHECK_ERROR()

	hipEventRecord(start, 0);

	// Run
	QueryKernel<BLOCK_THREADS, ITEMS_PER_THREAD>
	    <<<hardcoded.n_vec, BLOCK_THREADS>>>(base_col, bitpacked_col, hardcoded, d_result);
	CHECK_ERROR()

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time_query, start, stop);

	CubDebugExit(hipMemcpy(decoded_arr, d_result, 1024 * 1024 * 1024, hipMemcpyDeviceToHost));

	finish                             = chrono::high_resolution_clock::now();
	std::chrono::duration<double> diff = finish - st;

	double total_time_taken {diff.count() * 1000};
	FLS_SHOW(total_time_taken)

	/*Check the result*/

	for (size_t i {0}; i < 256 * 1024 * 1024; ++i) {
		if (org_col[i] != decoded_arr[i]) { throw std::runtime_error("RESULT INCOREECT!"); }
	}

	CLEANUP(d_result);

	return time_query;
}

n_t bitpacked_vec_n_tup(uint8_t bitdwith) {
	/**/
	return bitdwith * 32;
}

void shared_memory_delta_with_aggregation() {
	size_t         repeat           = 1;
	const uint64_t n_vec            = 256 * 1024;
	const uint64_t vec_sz           = 1024;
	const uint64_t n_tup            = vec_sz * n_vec;
	const uint64_t n_base           = 32 * n_vec;
	auto*          h_org_arr        = new uint32_t[n_tup];
	auto*          h_decoed_arr     = new uint32_t[n_tup];
	auto*          h_encoded_data   = new uint32_t[n_tup];
	auto*          h_transposed_arr = new uint32_t[vec_sz];
	auto*          h_unrsummed_arr  = new uint32_t[vec_sz];
	auto*          h_base_arr       = new uint32_t[n_base];
	uint64_t       encoded_arr_bsz  = n_tup * sizeof(int);
	uint32_t*      d_base_arr       = nullptr;
	uint32_t*      d_encoded_arr    = nullptr;

	for (uint8_t bitwidth {0}; bitwidth < 33; bitwidth++) {
		uint32_t bw = bitwidth;
		uint64_t sum {0};

		/* generate random numbers. */
		for (int i = 0; i < n_tup; i++) {
			if (bitwidth < 10) {
				h_org_arr[i] = bitwidth;
			}
		}

		FLS_SHOW(bw)

		auto in_als   = h_org_arr;
		auto out_als  = h_encoded_data;
		auto base_als = h_base_arr;

		for (uint64_t vec_idx {0}; vec_idx < n_vec; vec_idx++) {
			generated::transpose::fallback::scalar::transpose_i(in_als, h_transposed_arr);

			generated::unrsum::fallback::scalar::unrsum(h_transposed_arr, h_unrsummed_arr);

			std::memcpy(base_als, h_transposed_arr, sizeof(uint32_t) * 32);

			generated::pack::fallback::scalar::pack(h_unrsummed_arr, out_als, bitwidth);

			in_als   = in_als + vec_sz;
			out_als  = out_als + (bitwidth * 32);
			base_als = base_als + 32;
		}

		d_encoded_arr = load_to_gpu(h_encoded_data, encoded_arr_bsz, g_allocator);
		d_base_arr    = load_to_gpu(h_base_arr, 32 * n_vec * sizeof(uint32_t), g_allocator);

		CUDA_SAFE_CALL(hipDeviceSynchronize());

		QueryMtd query_mtd {n_vec, bitwidth, n_tup, sum};
		for (int i {0}; i < repeat; ++i) {
			auto time = decode<32, 32>(h_org_arr, h_decoed_arr, d_base_arr, d_encoded_arr, query_mtd, g_allocator);
			FLS_SHOW(time)
		}

		CLEANUP(d_encoded_arr)
		CLEANUP(d_base_arr)
	}
}

int main() {
	/**/
	shared_memory_delta_with_aggregation();
}