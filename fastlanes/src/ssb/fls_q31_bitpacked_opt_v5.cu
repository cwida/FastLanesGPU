#include "hip/hip_runtime.h"
// Ensure printing of CUDA runtime errors to console
#define HIPCUB_STDERR
#define SORTED

#include "crystal/crystal.cuh"
#include "crystal_ssb_utils.h"
#include "cub/test/test_util.h"
#include "fastlanes/join.cuh"
#include "fls_gen/unpack/unpack.cuh"
#include "gpu_utils.h"
#include "ssb_utils.h"
#include "gtest/gtest.h"
#include <cub/util_allocator.cuh>
#include <fls_gen/pack/pack.hpp>
#include <fls_gen/unpack/hardcoded_16.cuh>
#include <iostream>
#include <query/query_31.hpp>
#include <stdio.h>

using namespace std;
using namespace fastlanes::gpu;
using namespace fastlanes;

using namespace std;

auto query_mtd = ssb::ssb_q31_10;

template <int BLOCK_THREADS, int ITEMS_PER_THREAD>
__global__ void probe_v4(int* lo_orderdate,
                         int* lo_orderdate_bw,
                         int* lo_orderdate_base,
                         int* lo_orderdate_offset,
                         int* lo_custkey,
                         int* lo_suppkey,
                         int* lo_revenue,
                         int  lo_len,
                         int* ht_s,
                         int  s_len,
                         int* ht_c,
                         int  c_len,
                         int* ht_d,
                         int  d_len,
                         int* res) {

	int mtd_offset = blockIdx.x / 4;

	constexpr int TILE_SIZE = BLOCK_THREADS * ITEMS_PER_THREAD;
	// Load a segment of consecutive items that are blocked across threads
	int items[ITEMS_PER_THREAD];
	int selection_flags[ITEMS_PER_THREAD];
	int c_nation[ITEMS_PER_THREAD];
	int s_nation[ITEMS_PER_THREAD];
	int year[ITEMS_PER_THREAD];
	int revenue[ITEMS_PER_THREAD];

	int tile_offset    = blockIdx.x * TILE_SIZE;
	int num_tiles      = (lo_len + TILE_SIZE - 1) / TILE_SIZE;
	int num_tile_items = TILE_SIZE;

	if (blockIdx.x == num_tiles - 1) { num_tile_items = lo_len - tile_offset; }

	InitFlags<BLOCK_THREADS, ITEMS_PER_THREAD>(selection_flags);

	int suppkey_tile_offset = blockIdx.x * query_mtd.ssb.lo_chosen_suppkey_bw * 8;
	unpack_8_at_a_time::unpack_device(lo_suppkey + suppkey_tile_offset, items, query_mtd.ssb.lo_chosen_suppkey_bw);
	BlockProbeAndPHT_2<int, int, BLOCK_THREADS, ITEMS_PER_THREAD>(
	    items, s_nation, selection_flags, ht_s, s_len, num_tile_items);

	int custkey_tile_offset = blockIdx.x * query_mtd.ssb.lo_chosen_custkey_bw * 8;
	unpack_8_at_a_time::unpack_device(lo_custkey + custkey_tile_offset, items, query_mtd.ssb.lo_chosen_custkey_bw);
	BlockProbeAndPHT_2<int, int, BLOCK_THREADS, ITEMS_PER_THREAD>(
	    items, c_nation, selection_flags, ht_c, c_len, num_tile_items);

	int bw                    = lo_orderdate_bw[mtd_offset];
	int base                  = lo_orderdate_base[mtd_offset];
	int orderdate_tile_offset = lo_orderdate_offset[mtd_offset] + (blockIdx.x % 4) * bw * 8;

	unpack_8_at_a_time::unpack_device(lo_orderdate + orderdate_tile_offset, items, bw);
#pragma unroll
	for (int ITEM = 0; ITEM < ITEMS_PER_THREAD; ++ITEM) {
		items[ITEM] = items[ITEM] + base;
	}

	BlockProbeAndPHT_2<int, int, BLOCK_THREADS, ITEMS_PER_THREAD>(
	    items, year, selection_flags, ht_d, d_len, 0, num_tile_items);

	// BlockLoad<int, BLOCK_THREADS, ITEMS_PER_THREAD>(lo_revenue + tile_offset, revenue, num_tile_items);
	BlockPredLoad<int, int, BLOCK_THREADS, ITEMS_PER_THREAD>(
	    lo_revenue + tile_offset, revenue, num_tile_items, selection_flags);

#pragma unroll
	for (int ITEM = 0; ITEM < ITEMS_PER_THREAD; ++ITEM) {
		if ((threadIdx.x + (BLOCK_THREADS * ITEM)) < num_tile_items) {
			if (selection_flags[ITEM]) {
				int hash = (s_nation[ITEM] * 25 * 7 + c_nation[ITEM] * 7 + (year[ITEM] - 1992)) %
				           ((1998 - 1992 + 1) * 25 * 25);
				res[hash * 6]     = year[ITEM];
				res[hash * 6 + 1] = c_nation[ITEM];
				res[hash * 6 + 2] = s_nation[ITEM];
				/*atomicAdd(&res[hash * 6 + 4], revenue[ITEM]);*/
				atomicAdd(reinterpret_cast<unsigned long long*>(&res[hash * 6 + 4]), (long long)(revenue[ITEM]));
			}
		}
	}
}

template <int BLOCK_THREADS, int ITEMS_PER_THREAD>
__global__ void
build_hashtable_s(int* filter_col, int* dim_key, int* dim_val, int num_tuples, int* hash_table, int num_slots) {
	int TILE_SIZE = BLOCK_THREADS * ITEMS_PER_THREAD;

	int items[ITEMS_PER_THREAD];
	int items2[ITEMS_PER_THREAD];
	int selection_flags[ITEMS_PER_THREAD];

	int tile_offset    = blockIdx.x * TILE_SIZE;
	int num_tiles      = (num_tuples + TILE_SIZE - 1) / TILE_SIZE;
	int num_tile_items = TILE_SIZE;

	if (blockIdx.x == num_tiles - 1) { num_tile_items = num_tuples - tile_offset; }

	BlockLoad<int, BLOCK_THREADS, ITEMS_PER_THREAD>(filter_col + tile_offset, items, num_tile_items);
	BlockPredEQ<int, BLOCK_THREADS, ITEMS_PER_THREAD>(items, 2, selection_flags, num_tile_items);

	BlockLoad<int, BLOCK_THREADS, ITEMS_PER_THREAD>(dim_key + tile_offset, items, num_tile_items);
	BlockLoad<int, BLOCK_THREADS, ITEMS_PER_THREAD>(dim_val + tile_offset, items2, num_tile_items);
	BlockBuildSelectivePHT_2<int, int, BLOCK_THREADS, ITEMS_PER_THREAD>(
	    items, items2, selection_flags, hash_table, num_slots, num_tile_items);
}

template <int BLOCK_THREADS, int ITEMS_PER_THREAD>
__global__ void
build_hashtable_c(int* filter_col, int* dim_key, int* dim_val, int num_tuples, int* hash_table, int num_slots) {
	int TILE_SIZE = BLOCK_THREADS * ITEMS_PER_THREAD;

	int items[ITEMS_PER_THREAD];
	int items2[ITEMS_PER_THREAD];
	int selection_flags[ITEMS_PER_THREAD];

	int tile_offset    = blockIdx.x * TILE_SIZE;
	int num_tiles      = (num_tuples + TILE_SIZE - 1) / TILE_SIZE;
	int num_tile_items = TILE_SIZE;

	if (blockIdx.x == num_tiles - 1) { num_tile_items = num_tuples - tile_offset; }

	BlockLoad<int, BLOCK_THREADS, ITEMS_PER_THREAD>(filter_col + tile_offset, items, num_tile_items);
	BlockPredEQ<int, BLOCK_THREADS, ITEMS_PER_THREAD>(items, 2, selection_flags, num_tile_items);

	BlockLoad<int, BLOCK_THREADS, ITEMS_PER_THREAD>(dim_key + tile_offset, items, num_tile_items);
	BlockLoad<int, BLOCK_THREADS, ITEMS_PER_THREAD>(dim_val + tile_offset, items2, num_tile_items);
	BlockBuildSelectivePHT_2<int, int, BLOCK_THREADS, ITEMS_PER_THREAD>(
	    items, items2, selection_flags, hash_table, num_slots, num_tile_items);
}

template <int BLOCK_THREADS, int ITEMS_PER_THREAD>
__global__ void
build_hashtable_d(int* dim_key, int* dim_val, int num_tuples, int* hash_table, int num_slots, int val_min) {
	int TILE_SIZE = BLOCK_THREADS * ITEMS_PER_THREAD;

	int items[ITEMS_PER_THREAD];
	int items2[ITEMS_PER_THREAD];
	int selection_flags[ITEMS_PER_THREAD];

	int tile_offset    = blockIdx.x * TILE_SIZE;
	int num_tiles      = (num_tuples + TILE_SIZE - 1) / TILE_SIZE;
	int num_tile_items = TILE_SIZE;

	if (blockIdx.x == num_tiles - 1) { num_tile_items = num_tuples - tile_offset; }

	BlockLoad<int, BLOCK_THREADS, ITEMS_PER_THREAD>(dim_val + tile_offset, items, num_tile_items);
	BlockPredGTE<int, BLOCK_THREADS, ITEMS_PER_THREAD>(items, 1992, selection_flags, num_tile_items);
	BlockPredLTE<int, BLOCK_THREADS, ITEMS_PER_THREAD>(items, 1997, selection_flags, num_tile_items);

	BlockLoad<int, BLOCK_THREADS, ITEMS_PER_THREAD>(dim_key + tile_offset, items2, num_tile_items);
	BlockBuildSelectivePHT_2<int, int, BLOCK_THREADS, ITEMS_PER_THREAD>(
	    items2, items, selection_flags, hash_table, num_slots, 19920101, num_tile_items);
}

template <int BLOCK_THREADS, int ITEMS_PER_THREAD>
void runQuery(int*                         lo_orderdate,
              int*                         d_lo_orderdate_bw,
              int*                         d_lo_orderdate_base,
              int*                         d_lo_orderdate_offset,
              int*                         lo_custkey,
              int*                         lo_suppkey,
              int*                         lo_revenue,
              int                          lo_len,
              int*                         d_datekey,
              int*                         d_year,
              int                          d_len,
              int*                         s_suppkey,
              int*                         s_region,
              int*                         s_nation,
              int                          s_len,
              int*                         c_custkey,
              int*                         c_region,
              int*                         c_nation,
              int                          c_len,
              hipcub::CachingDeviceAllocator& g_allocator) {

	int *ht_d, *ht_c, *ht_s;
	int  d_val_len = 19981230 - 19920101 + 1;
	CubDebugExit(g_allocator.DeviceAllocate((void**)&ht_d, 2 * d_val_len * sizeof(int)));
	CubDebugExit(g_allocator.DeviceAllocate((void**)&ht_s, 2 * s_len * sizeof(int)));
	CubDebugExit(g_allocator.DeviceAllocate((void**)&ht_c, 2 * c_len * sizeof(int)));

	CubDebugExit(hipMemset(ht_d, 0, 2 * d_val_len * sizeof(int)));
	CubDebugExit(hipMemset(ht_s, 0, 2 * s_len * sizeof(int)));

	int tile_items = BLOCK_THREADS * ITEMS_PER_THREAD;
	build_hashtable_s<BLOCK_THREADS, ITEMS_PER_THREAD>
	    <<<(s_len + tile_items - 1) / tile_items, BLOCK_THREADS>>>(s_region, s_suppkey, s_nation, s_len, ht_s, s_len);
	/*CHECK_ERROR();*/

	build_hashtable_c<BLOCK_THREADS, ITEMS_PER_THREAD>
	    <<<(c_len + tile_items - 1) / tile_items, BLOCK_THREADS>>>(c_region, c_custkey, c_nation, c_len, ht_c, c_len);
	/*CHECK_ERROR();*/

	int d_val_min = 19920101;
	build_hashtable_d<BLOCK_THREADS, ITEMS_PER_THREAD><<<(d_len + tile_items - 1) / tile_items, BLOCK_THREADS>>>(
	    d_datekey, d_year, d_len, ht_d, d_val_len, d_val_min);
	/*CHECK_ERROR();*/

	int* res;
	int  res_size       = ((1998 - 1992 + 1) * 25 * 25);
	int  res_array_size = res_size * 6;
	CubDebugExit(g_allocator.DeviceAllocate((void**)&res, res_array_size * sizeof(int)));

	CubDebugExit(hipMemset(res, 0, res_array_size * sizeof(int)));

	// Run
	probe_v4<BLOCK_THREADS, ITEMS_PER_THREAD>
	    <<<(lo_len + tile_items - 1) / tile_items, BLOCK_THREADS>>>(lo_orderdate,
	                                                                d_lo_orderdate_bw,
	                                                                d_lo_orderdate_base,
	                                                                d_lo_orderdate_offset,
	                                                                lo_custkey,
	                                                                lo_suppkey,
	                                                                lo_revenue,
	                                                                lo_len,
	                                                                ht_s,
	                                                                s_len,
	                                                                ht_c,
	                                                                c_len,
	                                                                ht_d,
	                                                                d_val_len,
	                                                                res);

	int* h_res = new int[res_array_size];
	CubDebugExit(hipMemcpy(h_res, res, res_array_size * sizeof(int), hipMemcpyDeviceToHost));

	ssb::SSBQuery3ResultTable result_of_query;
	for (int i = 0; i < res_size; i++) {
		if (h_res[6 * i] != 0) {
			result_of_query.emplace_back(h_res[6 * i],
			                             h_res[6 * i + 1],
			                             h_res[6 * i + 2],
			                             reinterpret_cast<unsigned long long*>(&h_res[6 * i + 4])[0]);
		}
	}

	ASSERT_EQ(result_of_query.size(), ssb::ssb_q31_10.reuslt.size());
	ASSERT_EQ(result_of_query, ssb::ssb_q31_10.reuslt);

	delete[] h_res;
}

/**
 * Main
 */
int main(int argc, char* argv[]) {
	auto hard_coded     = query_mtd.ssb;
	int* h_lo_orderdate = loadColumn<int>("lo_orderdate", LO_LEN);
	int* h_lo_custkey   = loadColumn<int>("lo_custkey", LO_LEN);
	int* h_lo_suppkey   = loadColumn<int>("lo_suppkey", LO_LEN);
	int* h_lo_revenue   = loadColumn<int>("lo_revenue", LO_LEN);

	auto n_vec = hard_coded.n_vec;

	int* tmp = new int[n_vec * 1024];
	for (size_t i {0}; i < LO_LEN; ++i) {
		tmp[i] = h_lo_orderdate[i] - hard_coded.lo_orderdate_min;
	}

	const int* h_enc_lo_orderdate = new int[n_vec * 1024];
	const int* h_enc_lo_custkey   = new int[n_vec * 1024];
	const int* h_enc_lo_suppkey   = new int[n_vec * 1024];
	const int* h_enc_lo_revenue   = new int[n_vec * 1024];

	int* h_lo_orderdate_base   = new int[n_vec];
	int* h_lo_orderdate_bw     = new int[n_vec];
	int* h_lo_orderdate_offset = new int[n_vec];

	auto* orderdate_in = const_cast<int32_t*>(tmp);
	auto* custkey_in   = const_cast<int32_t*>(h_lo_custkey);
	auto* suppkey_in   = const_cast<int32_t*>(h_lo_suppkey);
	auto* revenue_in   = const_cast<int32_t*>(h_lo_revenue);

	auto* orderdate_out = const_cast<int32_t*>(h_enc_lo_orderdate);
	auto* custkey_out   = const_cast<int32_t*>(h_enc_lo_custkey);
	auto* suppkey_out   = const_cast<int32_t*>(h_enc_lo_suppkey);
	auto* revenue_out   = const_cast<int32_t*>(h_enc_lo_revenue);

	constexpr int SF10_LAST_VECTOR_IDX = 58580;
	constexpr int LAST_VECTOR_SIZE     = 294;
	for (uint64_t vec_idx {0}; vec_idx < n_vec; vec_idx++) {
		if (vec_idx == SF10_LAST_VECTOR_IDX) { set_zero_after<1024>(orderdate_in, LAST_VECTOR_SIZE); }

		h_lo_orderdate_base[vec_idx] = find_base<1024>(orderdate_in);
		subtract_base<1024>(orderdate_in, h_lo_orderdate_base[vec_idx]);
		h_lo_orderdate_bw[vec_idx] = find_bw<1024>(orderdate_in);

		if (vec_idx + 1 < n_vec) {
			h_lo_orderdate_offset[vec_idx + 1] = h_lo_orderdate_offset[vec_idx] + (h_lo_orderdate_bw[vec_idx] * 32);
		}

		if (h_lo_orderdate_bw[vec_idx] > 16) {
			std::cout << h_lo_orderdate_bw[vec_idx] << " bigger than 16 is not possible in orderdate! \n";
			exit(-2);
		}

		generated::pack::fallback::scalar::pack(orderdate_in, orderdate_out, h_lo_orderdate_bw[vec_idx]);
		orderdate_in  = orderdate_in + 1024;
		orderdate_out = orderdate_out + (h_lo_orderdate_bw[vec_idx] * 32);

		generated::pack::fallback::scalar::pack(custkey_in, custkey_out, hard_coded.lo_chosen_custkey_bw);
		custkey_in  = custkey_in + 1024;
		custkey_out = custkey_out + (hard_coded.lo_chosen_custkey_bw * 32);

		generated::pack::fallback::scalar::pack(suppkey_in, suppkey_out, hard_coded.lo_chosen_suppkey_bw);
		suppkey_in  = suppkey_in + 1024;
		suppkey_out = suppkey_out + (hard_coded.lo_chosen_suppkey_bw * 32);

		generated::pack::fallback::scalar::pack(revenue_in, revenue_out, hard_coded.lo_revenue_bw);
		revenue_in  = revenue_in + 1024;
		revenue_out = revenue_out + (hard_coded.lo_revenue_bw * 32);
	}

	int* d_lo_orderdate = loadToGPU<int32_t>(h_enc_lo_orderdate, hard_coded.n_tup_line_order, g_allocator);
	int* d_lo_custkey   = loadToGPU<int32_t>(h_enc_lo_custkey, hard_coded.n_tup_line_order, g_allocator);
	int* d_lo_suppkey   = loadToGPU<int32_t>(h_enc_lo_suppkey, hard_coded.n_tup_line_order, g_allocator);
	int* d_lo_revenue;

	d_lo_revenue = loadToGPU<int32_t>(h_lo_revenue, hard_coded.n_tup_line_order, g_allocator);

	int* h_d_datekey = loadColumn<int>("d_datekey", D_LEN);
	int* h_d_year    = loadColumn<int>("d_year", D_LEN);

	int* h_s_suppkey = loadColumn<int>("s_suppkey", S_LEN);
	int* h_s_nation  = loadColumn<int>("s_nation", S_LEN);
	int* h_s_region  = loadColumn<int>("s_region", S_LEN);

	int* h_c_custkey = loadColumn<int>("c_custkey", C_LEN);
	int* h_c_nation  = loadColumn<int>("c_nation", C_LEN);
	int* h_c_region  = loadColumn<int>("c_region", C_LEN);

	int* d_d_datekey = loadToGPU<int>(h_d_datekey, D_LEN, g_allocator);
	int* d_d_year    = loadToGPU<int>(h_d_year, D_LEN, g_allocator);

	int* d_s_suppkey = loadToGPU<int>(h_s_suppkey, S_LEN, g_allocator);
	int* d_s_region  = loadToGPU<int>(h_s_region, S_LEN, g_allocator);
	int* d_s_nation  = loadToGPU<int>(h_s_nation, S_LEN, g_allocator);

	int* d_c_custkey = loadToGPU<int>(h_c_custkey, C_LEN, g_allocator);
	int* d_c_region  = loadToGPU<int>(h_c_region, C_LEN, g_allocator);
	int* d_c_nation  = loadToGPU<int>(h_c_nation, C_LEN, g_allocator);

	int* d_lo_orderdate_base   = loadToGPU<int32_t>(h_lo_orderdate_base, n_vec, g_allocator);
	int* d_lo_orderdate_bw     = loadToGPU<int32_t>(h_lo_orderdate_bw, n_vec, g_allocator);
	int* d_lo_orderdate_offset = loadToGPU<int32_t>(h_lo_orderdate_offset, n_vec, g_allocator);

	// Run
	runQuery<32, 8>(d_lo_orderdate,
	                d_lo_orderdate_bw,
	                d_lo_orderdate_base,
	                d_lo_orderdate_offset,
	                d_lo_custkey,
	                d_lo_suppkey,
	                d_lo_revenue,
	                LO_LEN,
	                d_d_datekey,
	                d_d_year,
	                D_LEN,
	                d_s_suppkey,
	                d_s_region,
	                d_s_nation,
	                S_LEN,
	                d_c_custkey,
	                d_c_region,
	                d_c_nation,
	                C_LEN,
	                g_allocator);
}