#include "hip/hip_runtime.h"
// Ensure printing of CUDA runtime errors to console
#define HIPCUB_STDERR
#define SORTED

#include "crystal/crystal.cuh"
#include "crystal_ssb_utils.h"
#include "cub/test/test_util.h"
#include "fls_gen/unpack/unpack.cuh"
#include "gpu_utils.h"
#include "ssb_utils.h"
#include "gtest/gtest.h"
#include <cub/util_allocator.cuh>
#include <hip/hip_runtime.h>
#include <fls_gen/pack/pack.hpp>
#include <fls_gen/unpack/hardcoded_16.cuh>
#include <iostream>
#include <query/query_41.hpp>
#include <stdio.h>

using namespace std;
using namespace fastlanes::gpu;
using namespace fastlanes;

using namespace std;

auto query_mtd = ssb::ssb_q41_10;

template <int BLOCK_THREADS, int ITEMS_PER_THREAD>
__global__ void probe(int* lo_orderdate,
                      int* lo_orderdate_bw,
                      int* lo_orderdate_base,
                      int* lo_orderdate_offset,
                      int* lo_partkey,
                      int* lo_custkey,
                      int* lo_custkey_bw,
                      int* lo_custkey_base,
                      int* lo_custkey_offset,
                      int* lo_suppkey,
                      int* lo_revenue,
                      int* lo_supplycost,
                      int  lo_len,
                      int* ht_p,
                      int  p_len,
                      int* ht_s,
                      int  s_len,
                      int* ht_c,
                      int  c_len,
                      int* ht_d,
                      int  d_len,
                      int* res) {
	// Load a segment of consecutive items that are blocked across threads
	int TILE_SIZE = BLOCK_THREADS * ITEMS_PER_THREAD;

	int items[ITEMS_PER_THREAD];
	int selection_flags[ITEMS_PER_THREAD];
	int c_nation[ITEMS_PER_THREAD];
	// int s_nation[ITEMS_PER_THREAD];
	int year[ITEMS_PER_THREAD];
	int revenue[ITEMS_PER_THREAD];

	int tile_offset    = blockIdx.x * TILE_SIZE;
	int num_tiles      = (lo_len + TILE_SIZE - 1) / TILE_SIZE;
	int num_tile_items = TILE_SIZE;

	int mtd_offset = blockIdx.x / 4;

	if (blockIdx.x == num_tiles - 1) { num_tile_items = lo_len - tile_offset; }

	InitFlags<BLOCK_THREADS, ITEMS_PER_THREAD>(selection_flags);

	int suppkey_tile_offset = blockIdx.x * query_mtd.ssb.lo_chosen_suppkey_bw * 8;
	unpack_8_at_a_time::unpack_device(lo_suppkey + suppkey_tile_offset, items, query_mtd.ssb.lo_chosen_suppkey_bw);
	BlockProbeAndPHT_1<int, BLOCK_THREADS, ITEMS_PER_THREAD>(items, selection_flags, ht_s, s_len, num_tile_items);

	int bw                  = lo_custkey_bw[mtd_offset];
	int base                = lo_custkey_base[mtd_offset];
	int custkey_tile_offset = lo_custkey_offset[mtd_offset] + (blockIdx.x % 4) * bw * 8;
	unpack_8_at_a_time::unpack_device(lo_custkey + custkey_tile_offset, items, bw);
#pragma unroll
	for (int ITEM = 0; ITEM < ITEMS_PER_THREAD; ++ITEM) {
		items[ITEM] = items[ITEM] + base;
	}
	BlockProbeAndPHT_2<int, int, BLOCK_THREADS, ITEMS_PER_THREAD>(
	    items, c_nation, selection_flags, ht_c, c_len, num_tile_items);

	int partkey_tile_offset = blockIdx.x * query_mtd.ssb.lo_partkey_bw * 8;
	unpack_8_at_a_time::unpack_device(lo_partkey + partkey_tile_offset, items, query_mtd.ssb.lo_partkey_bw);
	BlockProbeAndPHT_1<int, BLOCK_THREADS, ITEMS_PER_THREAD>(items, selection_flags, ht_p, p_len, num_tile_items);

	bw                        = lo_orderdate_bw[mtd_offset];
	base                      = lo_orderdate_base[mtd_offset];
	int orderdate_tile_offset = lo_orderdate_offset[mtd_offset] + (blockIdx.x % 4) * bw * 8;

	unpack_8_at_a_time::unpack_device(lo_orderdate + orderdate_tile_offset, items, bw);
#pragma unroll
	for (int ITEM = 0; ITEM < ITEMS_PER_THREAD; ++ITEM) {
		items[ITEM] = items[ITEM] + base;
	}
	BlockProbeAndPHT_2<int, int, BLOCK_THREADS, ITEMS_PER_THREAD>(
	    items, year, selection_flags, ht_d, d_len, 0, num_tile_items);

	BlockPredLoad<int, int, BLOCK_THREADS, ITEMS_PER_THREAD>(
	    lo_revenue + tile_offset, revenue, num_tile_items, selection_flags);

	int supplycost_tile_offset = blockIdx.x * query_mtd.ssb.lo_chosen_supplycost_bw * 8;
	unpack_8_at_a_time::unpack_device(
	    lo_supplycost + supplycost_tile_offset, items, query_mtd.ssb.lo_chosen_supplycost_bw);

#pragma unroll
	for (int ITEM = 0; ITEM < ITEMS_PER_THREAD; ++ITEM) {
		if (threadIdx.x + (BLOCK_THREADS * ITEM) < num_tile_items) {
			if (selection_flags[ITEM]) {
				int hash          = (c_nation[ITEM] * 7 + (year[ITEM] - 1992)) % ((1998 - 1992 + 1) * 25);
				res[hash * 4]     = year[ITEM];
				res[hash * 4 + 1] = c_nation[ITEM];
				/*atomicAdd(&res[hash * 4 + 2], (1));*/
				/*atomicAdd(reinterpret_cast<unsigned long long*>(&res[hash * 4 + 2]), (long long)(1));*/
				atomicAdd(reinterpret_cast<unsigned long long*>(&res[hash * 4 + 2]),
				          (long long)(revenue[ITEM] - items[ITEM]));
			}
		}
	}
}

template <int BLOCK_THREADS, int ITEMS_PER_THREAD>
__global__ void build_hashtable_s(int* filter_col, int* dim_key, int num_tuples, int* hash_table, int num_slots) {
	int TILE_SIZE = BLOCK_THREADS * ITEMS_PER_THREAD;

	int items[ITEMS_PER_THREAD];
	int selection_flags[ITEMS_PER_THREAD];

	int tile_offset    = blockIdx.x * TILE_SIZE;
	int num_tiles      = (num_tuples + TILE_SIZE - 1) / TILE_SIZE;
	int num_tile_items = TILE_SIZE;

	if (blockIdx.x == num_tiles - 1) { num_tile_items = num_tuples - tile_offset; }

	BlockLoad<int, BLOCK_THREADS, ITEMS_PER_THREAD>(filter_col + tile_offset, items, num_tile_items);
	BlockPredEQ<int, BLOCK_THREADS, ITEMS_PER_THREAD>(items, 1, selection_flags, num_tile_items);

	BlockLoad<int, BLOCK_THREADS, ITEMS_PER_THREAD>(dim_key + tile_offset, items, num_tile_items);
	BlockBuildSelectivePHT_1<int, BLOCK_THREADS, ITEMS_PER_THREAD>(
	    items, selection_flags, hash_table, num_slots, num_tile_items);
}

template <int BLOCK_THREADS, int ITEMS_PER_THREAD>
__global__ void build_hashtable_p(int* filter_col, int* dim_key, int num_tuples, int* hash_table, int num_slots) {
	int TILE_SIZE = BLOCK_THREADS * ITEMS_PER_THREAD;

	int items[ITEMS_PER_THREAD];
	int selection_flags[ITEMS_PER_THREAD];

	int tile_offset    = blockIdx.x * TILE_SIZE;
	int num_tiles      = (num_tuples + TILE_SIZE - 1) / TILE_SIZE;
	int num_tile_items = TILE_SIZE;

	if (blockIdx.x == num_tiles - 1) { num_tile_items = num_tuples - tile_offset; }

	BlockLoad<int, BLOCK_THREADS, ITEMS_PER_THREAD>(filter_col + tile_offset, items, num_tile_items);
	BlockPredEQ<int, BLOCK_THREADS, ITEMS_PER_THREAD>(items, 0, selection_flags, num_tile_items);
	BlockPredOrEQ<int, BLOCK_THREADS, ITEMS_PER_THREAD>(items, 1, selection_flags, num_tile_items);

	BlockLoad<int, BLOCK_THREADS, ITEMS_PER_THREAD>(dim_key + tile_offset, items, num_tile_items);
	BlockBuildSelectivePHT_1<int, BLOCK_THREADS, ITEMS_PER_THREAD>(
	    items, selection_flags, hash_table, num_slots, num_tile_items);
}

template <int BLOCK_THREADS, int ITEMS_PER_THREAD>
__global__ void
build_hashtable_c(int* filter_col, int* dim_key, int* dim_val, int num_tuples, int* hash_table, int num_slots) {
	int TILE_SIZE = BLOCK_THREADS * ITEMS_PER_THREAD;

	int items[ITEMS_PER_THREAD];
	int items2[ITEMS_PER_THREAD];
	int selection_flags[ITEMS_PER_THREAD];

	int tile_offset    = blockIdx.x * TILE_SIZE;
	int num_tiles      = (num_tuples + TILE_SIZE - 1) / TILE_SIZE;
	int num_tile_items = TILE_SIZE;

	if (blockIdx.x == num_tiles - 1) { num_tile_items = num_tuples - tile_offset; }

	BlockLoad<int, BLOCK_THREADS, ITEMS_PER_THREAD>(filter_col + tile_offset, items, num_tile_items);
	BlockPredEQ<int, BLOCK_THREADS, ITEMS_PER_THREAD>(items, 1, selection_flags, num_tile_items);

	BlockLoad<int, BLOCK_THREADS, ITEMS_PER_THREAD>(dim_key + tile_offset, items, num_tile_items);
	BlockLoad<int, BLOCK_THREADS, ITEMS_PER_THREAD>(dim_val + tile_offset, items2, num_tile_items);
	BlockBuildSelectivePHT_2<int, int, BLOCK_THREADS, ITEMS_PER_THREAD>(
	    items, items2, selection_flags, hash_table, num_slots, num_tile_items);
}

template <int BLOCK_THREADS, int ITEMS_PER_THREAD>
__global__ void
build_hashtable_d(int* dim_key, int* dim_val, int num_tuples, int* hash_table, int num_slots, int val_min) {
	int TILE_SIZE = BLOCK_THREADS * ITEMS_PER_THREAD;

	int items[ITEMS_PER_THREAD];
	int items2[ITEMS_PER_THREAD];
	int selection_flags[ITEMS_PER_THREAD];

	int tile_offset    = blockIdx.x * TILE_SIZE;
	int num_tiles      = (num_tuples + TILE_SIZE - 1) / TILE_SIZE;
	int num_tile_items = TILE_SIZE;

	if (blockIdx.x == num_tiles - 1) { num_tile_items = num_tuples - tile_offset; }

	InitFlags<BLOCK_THREADS, ITEMS_PER_THREAD>(selection_flags);
	BlockLoad<int, BLOCK_THREADS, ITEMS_PER_THREAD>(dim_key + tile_offset, items, num_tile_items);
	BlockLoad<int, BLOCK_THREADS, ITEMS_PER_THREAD>(dim_val + tile_offset, items2, num_tile_items);
	BlockBuildSelectivePHT_2<int, int, BLOCK_THREADS, ITEMS_PER_THREAD>(
	    items, items2, selection_flags, hash_table, num_slots, val_min, num_tile_items);
}

template <int BLOCK_THREADS, int ITEMS_PER_THREAD>
void runQuery(int*                         lo_orderdate,
              int*                         lo_orderdate_bw,
              int*                         lo_orderdate_base,
              int*                         lo_orderdate_offset,
              int*                         lo_custkey,
              int*                         lo_custkey_bw,
              int*                         lo_custkey_base,
              int*                         lo_custkey_offset,
              int*                         lo_partkey,
              int*                         lo_suppkey,
              int*                         lo_revenue,
              int*                         lo_supplycost,
              int                          lo_len,
              int*                         d_datekey,
              int*                         d_year,
              int                          d_len,
              int*                         p_partkey,
              int*                         p_mfgr,
              int                          p_len,
              int*                         s_suppkey,
              int*                         s_region,
              int                          s_len,
              int*                         c_custkey,
              int*                         c_region,
              int*                         c_nation,
              int                          c_len,
              hipcub::CachingDeviceAllocator& g_allocator) {
	SETUP_TIMING();

	float time_query;

	hipEventRecord(start, 0);

	int *ht_d, *ht_c, *ht_s, *ht_p;
	int  d_val_len = 19981230 - 19920101 + 1;
	CubDebugExit(g_allocator.DeviceAllocate((void**)&ht_d, 2 * d_val_len * sizeof(int)));
	CubDebugExit(g_allocator.DeviceAllocate((void**)&ht_s, 2 * s_len * sizeof(int)));
	CubDebugExit(g_allocator.DeviceAllocate((void**)&ht_c, 2 * c_len * sizeof(int)));
	CubDebugExit(g_allocator.DeviceAllocate((void**)&ht_p, 2 * p_len * sizeof(int)));

	CubDebugExit(hipMemset(ht_d, 0, 2 * d_val_len * sizeof(int)));
	CubDebugExit(hipMemset(ht_s, 0, 2 * s_len * sizeof(int)));
	CubDebugExit(hipMemset(ht_c, 0, 2 * c_len * sizeof(int)));
	CubDebugExit(hipMemset(ht_p, 0, 2 * p_len * sizeof(int)));

	int tile_items = BLOCK_THREADS * ITEMS_PER_THREAD;
	build_hashtable_s<BLOCK_THREADS, ITEMS_PER_THREAD>
	    <<<(s_len + tile_items - 1) / tile_items, BLOCK_THREADS>>>(s_region, s_suppkey, s_len, ht_s, s_len);
	/*CHECK_ERROR();*/

	int* s_res = new int[s_len * 2];
	CubDebugExit(hipMemcpy(s_res, ht_s, s_len * 2 * sizeof(int), hipMemcpyDeviceToHost));

	build_hashtable_c<BLOCK_THREADS, ITEMS_PER_THREAD>
	    <<<(c_len + tile_items - 1) / tile_items, BLOCK_THREADS>>>(c_region, c_custkey, c_nation, c_len, ht_c, c_len);
	/*CHECK_ERROR();*/

	int* c_res = new int[c_len * 2];
	CubDebugExit(hipMemcpy(c_res, ht_c, c_len * 2 * sizeof(int), hipMemcpyDeviceToHost));

	build_hashtable_p<BLOCK_THREADS, ITEMS_PER_THREAD>
	    <<<(p_len + tile_items - 1) / tile_items, BLOCK_THREADS>>>(p_mfgr, p_partkey, p_len, ht_p, p_len);
	/*CHECK_ERROR();*/

	int* p_res = new int[p_len * 2];
	CubDebugExit(hipMemcpy(p_res, ht_p, p_len * 2 * sizeof(int), hipMemcpyDeviceToHost));

	int d_val_min = 19920101;
	build_hashtable_d<BLOCK_THREADS, ITEMS_PER_THREAD><<<(d_len + tile_items - 1) / tile_items, BLOCK_THREADS>>>(
	    d_datekey, d_year, d_len, ht_d, d_val_len, d_val_min);
	/*CHECK_ERROR();*/

	int* res;
	int  res_size       = ((1998 - 1992 + 1) * 25);
	int  ht_entries     = 4; // int,int,long long
	int  res_array_size = res_size * ht_entries;
	CubDebugExit(g_allocator.DeviceAllocate((void**)&res, res_array_size * sizeof(int)));

	CubDebugExit(hipMemset(res, 0, res_array_size * sizeof(int)));

	// Run
	probe<BLOCK_THREADS, ITEMS_PER_THREAD>
	    <<<(lo_len + tile_items - 1) / tile_items, BLOCK_THREADS>>>(lo_orderdate,
	                                                                lo_orderdate_bw,
	                                                                lo_orderdate_base,
	                                                                lo_orderdate_offset,
	                                                                lo_partkey,
	                                                                lo_custkey,
	                                                                lo_custkey_bw,
	                                                                lo_custkey_base,
	                                                                lo_custkey_offset,
	                                                                lo_suppkey,
	                                                                lo_revenue,
	                                                                lo_supplycost,
	                                                                lo_len,
	                                                                ht_p,
	                                                                p_len,
	                                                                ht_s,
	                                                                s_len,
	                                                                ht_c,
	                                                                c_len,
	                                                                ht_d,
	                                                                d_val_len,
	                                                                res);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time_query, start, stop);

	int* h_res = new int[res_array_size];
	CubDebugExit(hipMemcpy(h_res, res, res_array_size * sizeof(int), hipMemcpyDeviceToHost));

	// cout << "Result:" << endl;
	// int res_count = 0;
	// for (int i = 0; i < res_size; i++) {
	// 	if (h_res[4 * i] != 0) {
	// 		cout << h_res[4 * i] << " " << h_res[4 * i + 1] << " "
	// 		     << reinterpret_cast<unsigned long long*>(&h_res[4 * i + 2])[0] << endl;
	// 		res_count += 1;
	// 	}
	// }
	//
	// cout << "Res Count: " << res_count << endl;

	ssb::SSBQuery4ResultTable result_of_query;
	for (int i = 0; i < res_size; i++) {
		if (h_res[4 * i] != 0) {
			result_of_query.emplace_back(
			    h_res[4 * i], h_res[4 * i + 1], reinterpret_cast<unsigned long long*>(&h_res[4 * i + 2])[0]);
		}
	}

	ASSERT_EQ(result_of_query.size(), ssb::ssb_q41_10.reuslt.size());
	ASSERT_EQ(result_of_query, ssb::ssb_q41_10.reuslt);

	delete[] h_res;
}

/**
 * Main
 */
int main() {
	auto hard_coded      = query_mtd.ssb;
	int* h_lo_orderdate  = loadColumn<int>("lo_orderdate", LO_LEN);
	int* h_lo_suppkey    = loadColumn<int>("lo_suppkey", LO_LEN);
	int* h_lo_custkey    = loadColumn<int>("lo_custkey", LO_LEN);
	int* h_lo_partkey    = loadColumn<int>("lo_partkey", LO_LEN);
	int* h_lo_revenue    = loadColumn<int>("lo_revenue", LO_LEN);
	int* h_lo_supplycost = loadColumn<int>("lo_supplycost", LO_LEN);

	auto n_vec = hard_coded.n_vec;

	int* tmp = new int[n_vec * 1024];
	for (size_t i {0}; i < LO_LEN; ++i) {
		tmp[i] = h_lo_orderdate[i] - hard_coded.lo_orderdate_min;
	}

	std::cout << "h_lo_orderdate | " << std::boolalpha << is_sorted(h_lo_orderdate, LO_LEN) << "\n";

	int* h_lo_orderdate_base   = new int[n_vec];
	int* h_lo_orderdate_bw     = new int[n_vec];
	int* h_lo_orderdate_offset = new int[n_vec];

	const int* h_enc_lo_orderdate  = new int[n_vec * 1024];
	int*       h_lo_custkey_base   = new int[n_vec];
	int*       h_lo_custkey_bw     = new int[n_vec];
	int*       h_lo_custkey_offset = new int[n_vec];

	const int* h_enc_lo_custkey    = new int[n_vec * 1024];
	const int* h_enc_lo_suppkey    = new int[n_vec * 1024];
	const int* h_enc_lo_revenue    = new int[n_vec * 1024];
	const int* h_enc_lo_partkey    = new int[n_vec * 1024];
	const int* h_enc_lo_supplycost = new int[n_vec * 1024];

	auto* orderdate_in  = const_cast<int32_t*>(tmp);
	auto* custkey_in    = const_cast<int32_t*>(h_lo_custkey);
	auto* suppkey_in    = const_cast<int32_t*>(h_lo_suppkey);
	auto* revenue_in    = const_cast<int32_t*>(h_lo_revenue);
	auto* partkey_in    = const_cast<int32_t*>(h_lo_partkey);
	auto* supplycost_in = const_cast<int32_t*>(h_lo_supplycost);

	auto* orderdate_out  = const_cast<int32_t*>(h_enc_lo_orderdate);
	auto* custkey_out    = const_cast<int32_t*>(h_enc_lo_custkey);
	auto* suppkey_out    = const_cast<int32_t*>(h_enc_lo_suppkey);
	auto* revenue_out    = const_cast<int32_t*>(h_enc_lo_revenue);
	auto* partkey_out    = const_cast<int32_t*>(h_enc_lo_partkey);
	auto* supplycost_out = const_cast<int32_t*>(h_enc_lo_supplycost);

	constexpr int SF10_LAST_VECTOR_IDX = 58580;
	constexpr int LAST_VECTOR_SIZE     = 294;

	h_lo_orderdate_offset[0] = 0;
	for (uint64_t vec_idx {0}; vec_idx < n_vec; vec_idx++) {
		if (vec_idx == SF10_LAST_VECTOR_IDX) { set_zero_after<1024>(custkey_in, LAST_VECTOR_SIZE); }
		if (vec_idx == SF10_LAST_VECTOR_IDX) { set_zero_after<1024>(orderdate_in, LAST_VECTOR_SIZE); }

		h_lo_orderdate_base[vec_idx] = find_base<1024>(orderdate_in);
		subtract_base<1024>(orderdate_in, h_lo_orderdate_base[vec_idx]);
		h_lo_orderdate_bw[vec_idx] = find_bw<1024>(orderdate_in);

		if (vec_idx + 1 < n_vec) {
			h_lo_orderdate_offset[vec_idx + 1] = h_lo_orderdate_offset[vec_idx] + (h_lo_orderdate_bw[vec_idx] * 32);
		}

		if (h_lo_orderdate_bw[vec_idx] > 16) {
			std::cout << h_lo_orderdate_bw[vec_idx] << " bigger than 16 is not possible in orderdate! \n";
			exit(-2);
		}

		generated::pack::fallback::scalar::pack(orderdate_in, orderdate_out, h_lo_orderdate_bw[vec_idx]);
		orderdate_in  = orderdate_in + 1024;
		orderdate_out = orderdate_out + (h_lo_orderdate_bw[vec_idx] * 32);

		generated::pack::fallback::scalar::pack(partkey_in, partkey_out, hard_coded.lo_partkey_bw);
		partkey_in  = partkey_in + 1024;
		partkey_out = partkey_out + (hard_coded.lo_partkey_bw * 32);

		generated::pack::fallback::scalar::pack(supplycost_in, supplycost_out, hard_coded.lo_chosen_supplycost_bw);
		supplycost_in  = supplycost_in + 1024;
		supplycost_out = supplycost_out + (hard_coded.lo_chosen_supplycost_bw * 32);

		h_lo_custkey_base[vec_idx] = find_base<1024>(custkey_in);
		subtract_base<1024>(custkey_in, h_lo_custkey_base[vec_idx]);
		h_lo_custkey_bw[vec_idx] = find_bw<1024>(custkey_in);

		if (vec_idx + 1 < n_vec) {
			h_lo_custkey_offset[vec_idx + 1] = h_lo_custkey_offset[vec_idx] + (h_lo_custkey_bw[vec_idx] * 32);
		}

		if (h_lo_custkey_bw[vec_idx] > 20) {
			std::cout << vec_idx << std::endl;
			std::cout << h_lo_custkey_bw[vec_idx] << "   bigger than 20 is not possible in custkey! \n";
			exit(-2);
		}

		generated::pack::fallback::scalar::pack(custkey_in, custkey_out, h_lo_custkey_bw[vec_idx]);
		custkey_in  = custkey_in + 1024;
		custkey_out = custkey_out + (h_lo_custkey_bw[vec_idx] * 32);

		generated::pack::fallback::scalar::pack(suppkey_in, suppkey_out, hard_coded.lo_chosen_suppkey_bw);
		suppkey_in  = suppkey_in + 1024;
		suppkey_out = suppkey_out + (hard_coded.lo_chosen_suppkey_bw * 32);

		generated::pack::fallback::scalar::pack(revenue_in, revenue_out, hard_coded.lo_revenue_bw);
		revenue_in  = revenue_in + 1024;
		revenue_out = revenue_out + (hard_coded.lo_revenue_bw * 32);
	}

	int* d_lo_orderdate  = loadToGPU<int32_t>(h_enc_lo_orderdate, hard_coded.n_tup_line_order, g_allocator);
	int* d_lo_custkey    = loadToGPU<int32_t>(h_enc_lo_custkey, hard_coded.n_tup_line_order, g_allocator);
	int* d_lo_suppkey    = loadToGPU<int32_t>(h_enc_lo_suppkey, hard_coded.n_tup_line_order, g_allocator);
	int* d_lo_revenue    = loadToGPU<int32_t>(h_lo_revenue, hard_coded.n_tup_line_order, g_allocator);
	int* d_lo_partkey    = loadToGPU<int32_t>(h_enc_lo_partkey, hard_coded.n_tup_line_order, g_allocator);
	int* d_lo_supplycost = loadToGPU<int32_t>(h_enc_lo_supplycost, hard_coded.n_tup_line_order, g_allocator);

	int* h_d_datekey      = loadColumn<int>("d_datekey", D_LEN);
	int* h_d_year         = loadColumn<int>("d_year", D_LEN);
	int* h_d_yearmonthnum = loadColumn<int>("d_yearmonthnum", D_LEN);

	int* h_s_suppkey = loadColumn<int>("s_suppkey", S_LEN);
	int* h_s_region  = loadColumn<int>("s_region", S_LEN);

	int* h_p_partkey = loadColumn<int>("p_partkey", P_LEN);
	int* h_p_mfgr    = loadColumn<int>("p_mfgr", P_LEN);

	int* h_c_custkey = loadColumn<int>("c_custkey", C_LEN);
	int* h_c_region  = loadColumn<int>("c_region", C_LEN);
	int* h_c_nation  = loadColumn<int>("c_nation", C_LEN);

	cout << "** LOADED DATA **" << endl;

	int* d_lo_orderdate_base   = loadToGPU<int32_t>(h_lo_orderdate_base, n_vec, g_allocator);
	int* d_lo_orderdate_bw     = loadToGPU<int32_t>(h_lo_orderdate_bw, n_vec, g_allocator);
	int* d_lo_orderdate_offset = loadToGPU<int32_t>(h_lo_orderdate_offset, n_vec, g_allocator);

	int* d_lo_custkey_base   = loadToGPU<int32_t>(h_lo_custkey_base, n_vec, g_allocator);
	int* d_lo_custkey_bw     = loadToGPU<int32_t>(h_lo_custkey_bw, n_vec, g_allocator);
	int* d_lo_custkey_offset = loadToGPU<int32_t>(h_lo_custkey_offset, n_vec, g_allocator);

	int* d_d_datekey = loadToGPU<int>(h_d_datekey, D_LEN, g_allocator);
	int* d_d_year    = loadToGPU<int>(h_d_year, D_LEN, g_allocator);

	int* d_p_partkey = loadToGPU<int>(h_p_partkey, P_LEN, g_allocator);
	int* d_p_mfgr    = loadToGPU<int>(h_p_mfgr, P_LEN, g_allocator);

	int* d_s_suppkey = loadToGPU<int>(h_s_suppkey, S_LEN, g_allocator);
	int* d_s_region  = loadToGPU<int>(h_s_region, S_LEN, g_allocator);

	int* d_c_custkey = loadToGPU<int>(h_c_custkey, C_LEN, g_allocator);
	int* d_c_region  = loadToGPU<int>(h_c_region, C_LEN, g_allocator);
	int* d_c_nation  = loadToGPU<int>(h_c_nation, C_LEN, g_allocator);

	cout << "** LOADED DATA TO GPU **" << endl;

	runQuery<32, 8>(d_lo_orderdate,
	                d_lo_orderdate_bw,
	                d_lo_orderdate_base,
	                d_lo_orderdate_offset,
	                d_lo_custkey,
	                d_lo_custkey_bw,
	                d_lo_custkey_base,
	                d_lo_custkey_offset,
	                d_lo_partkey,
	                d_lo_suppkey,
	                d_lo_revenue,
	                d_lo_supplycost,
	                LO_LEN,
	                d_d_datekey,
	                d_d_year,
	                D_LEN,
	                d_p_partkey,
	                d_p_mfgr,
	                P_LEN,
	                d_s_suppkey,
	                d_s_region,
	                S_LEN,
	                d_c_custkey,
	                d_c_region,
	                d_c_nation,
	                C_LEN,
	                g_allocator);

	return 0;
}
