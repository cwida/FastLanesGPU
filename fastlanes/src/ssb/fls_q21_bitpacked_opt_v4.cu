#include "hip/hip_runtime.h"
// Ensure printing of CUDA runtime errors to console
#define HIPCUB_STDERR
#define SORTED

#include "crystal/crystal.cuh"
#include "crystal_ssb_utils.h"
#include "cub/test/test_util.h"
#include "fls_gen/pack/pack.hpp"
#include "fls_gen/unpack/hardcoded_16.cuh"
#include "fls_gen/unpack/unpack_fused.cuh"
#include "gpu_utils.h"
#include "query/query_21.hpp"
#include "ssb_utils.h"
#include "gtest/gtest.h"
#include <cub/util_allocator.cuh>
#include <iostream>
#include <vector>

using namespace std;
using namespace fastlanes::gpu;
using namespace fastlanes;

inline auto query_mtd = ssb::ssb_q21_10;

template <int BLOCK_THREADS, int ITEMS_PER_THREAD>
__global__ void probe_v3(int* lo_orderdate,
                         int* lo_orderdate_bw,
                         int* lo_orderdate_base,
                         int* lo_orderdate_offset,
                         int* lo_partkey,
                         int* lo_suppkey,
                         int* lo_revenue,
                         int  lo_len,
                         int* ht_s,
                         int  s_len,
                         int* ht_p,
                         int  p_len,
                         int* ht_d,
                         int  d_len,
                         int* res) {
	int mtd_offset = blockIdx.x / 4;

	int TILE_SIZE = BLOCK_THREADS * ITEMS_PER_THREAD;
	// Load a tile striped across threads
	int items[ITEMS_PER_THREAD];
	int selection_flags[ITEMS_PER_THREAD];
	int brand[ITEMS_PER_THREAD];
	int year[ITEMS_PER_THREAD];
	int revenue[ITEMS_PER_THREAD];

	int tile_offset    = blockIdx.x * TILE_SIZE;
	int num_tiles      = (lo_len + TILE_SIZE - 1) / TILE_SIZE;
	int num_tile_items = TILE_SIZE;

	if (blockIdx.x == num_tiles - 1) { num_tile_items = lo_len - tile_offset; }

	InitFlags<BLOCK_THREADS, ITEMS_PER_THREAD>(selection_flags);

	int suppkey_tile_offset = blockIdx.x * query_mtd.ssb.lo_chosen_suppkey_bw * 8;
	unpack_8_at_a_time::unpack_device(lo_suppkey + suppkey_tile_offset, items, query_mtd.ssb.lo_chosen_suppkey_bw);
	BlockProbeAndPHT_1<int, BLOCK_THREADS, ITEMS_PER_THREAD>(items, selection_flags, ht_s, s_len, num_tile_items);

	int partkey_tile_offset = blockIdx.x * query_mtd.ssb.lo_partkey_bw * 8;
	unpack_8_at_a_time::unpack_device(lo_partkey + partkey_tile_offset, items, query_mtd.ssb.lo_partkey_bw);
	BlockProbeAndPHT_2<int, int, BLOCK_THREADS, ITEMS_PER_THREAD>(
	    items, brand, selection_flags, ht_p, p_len, num_tile_items);

	int bw                    = lo_orderdate_bw[mtd_offset];
	int base                  = lo_orderdate_base[mtd_offset];
	int orderdate_tile_offset = lo_orderdate_offset[mtd_offset] + (blockIdx.x % 4) * bw * 8;

	unpack_8_at_a_time::unpack_device(lo_orderdate + orderdate_tile_offset, items, bw);
#pragma unroll
	for (int ITEM = 0; ITEM < ITEMS_PER_THREAD; ++ITEM) {
		items[ITEM] = items[ITEM] + base;
	}

	BlockProbeAndPHT_2<int, int, BLOCK_THREADS, ITEMS_PER_THREAD>(
	    items, year, selection_flags, ht_d, d_len, 0, num_tile_items);

	BlockPredLoad<int, int, BLOCK_THREADS, ITEMS_PER_THREAD>(
	    lo_revenue + tile_offset, revenue, num_tile_items, selection_flags);

#pragma unroll
	for (int ITEM = 0; ITEM < ITEMS_PER_THREAD; ++ITEM) {
		if ((threadIdx.x + (BLOCK_THREADS * ITEM)) < num_tile_items) {
			if (selection_flags[ITEM]) {
				int hash          = (brand[ITEM] * 7 + (year[ITEM] - 1992)) % ((1998 - 1992 + 1) * (5 * 5 * 40));
				res[hash * 4]     = year[ITEM];
				res[hash * 4 + 1] = brand[ITEM];
				atomicAdd(reinterpret_cast<unsigned long long*>(&res[hash * 4 + 2]), (long long)(revenue[ITEM]));
			}
		}
	}
}

template <int BLOCK_THREADS, int ITEMS_PER_THREAD>
__global__ void build_hashtable_s(int* filter_col, int* dim_key, int num_tuples, int* hash_table, int num_slots) {
	int TILE_SIZE = BLOCK_THREADS * ITEMS_PER_THREAD;

	int items[ITEMS_PER_THREAD];
	int selection_flags[ITEMS_PER_THREAD];

	int tile_offset    = blockIdx.x * TILE_SIZE;
	int num_tiles      = (num_tuples + TILE_SIZE - 1) / TILE_SIZE;
	int num_tile_items = TILE_SIZE;

	if (blockIdx.x == num_tiles - 1) { num_tile_items = num_tuples - tile_offset; }

	BlockLoad<int, BLOCK_THREADS, ITEMS_PER_THREAD>(filter_col + tile_offset, items, num_tile_items);
	BlockPredEQ<int, BLOCK_THREADS, ITEMS_PER_THREAD>(items, 1, selection_flags, num_tile_items);

	BlockLoad<int, BLOCK_THREADS, ITEMS_PER_THREAD>(dim_key + tile_offset, items, num_tile_items);
	BlockBuildSelectivePHT_1<int, BLOCK_THREADS, ITEMS_PER_THREAD>(
	    items, selection_flags, hash_table, num_slots, num_tile_items);
}

template <int BLOCK_THREADS, int ITEMS_PER_THREAD>
__global__ void
build_hashtable_p(int* filter_col, int* dim_key, int* dim_val, int num_tuples, int* hash_table, int num_slots) {
	int TILE_SIZE = BLOCK_THREADS * ITEMS_PER_THREAD;

	int items[ITEMS_PER_THREAD];
	int items2[ITEMS_PER_THREAD];
	int selection_flags[ITEMS_PER_THREAD];

	int tile_offset    = blockIdx.x * TILE_SIZE;
	int num_tiles      = (num_tuples + TILE_SIZE - 1) / TILE_SIZE;
	int num_tile_items = TILE_SIZE;

	if (blockIdx.x == num_tiles - 1) { num_tile_items = num_tuples - tile_offset; }

	BlockLoad<int, BLOCK_THREADS, ITEMS_PER_THREAD>(filter_col + tile_offset, items, num_tile_items);
	BlockPredEQ<int, BLOCK_THREADS, ITEMS_PER_THREAD>(items, 1, selection_flags, num_tile_items);

	BlockLoad<int, BLOCK_THREADS, ITEMS_PER_THREAD>(dim_key + tile_offset, items, num_tile_items);
	BlockLoad<int, BLOCK_THREADS, ITEMS_PER_THREAD>(dim_val + tile_offset, items2, num_tile_items);
	BlockBuildSelectivePHT_2<int, int, BLOCK_THREADS, ITEMS_PER_THREAD>(
	    items, items2, selection_flags, hash_table, num_slots, num_tile_items);
}

template <int BLOCK_THREADS, int ITEMS_PER_THREAD>
__global__ void
build_hashtable_d(int* dim_key, int* dim_val, int num_tuples, int* hash_table, int num_slots, int val_min) {
	int TILE_SIZE = BLOCK_THREADS * ITEMS_PER_THREAD;

	int items[ITEMS_PER_THREAD];
	int items2[ITEMS_PER_THREAD];
	int selection_flags[ITEMS_PER_THREAD];

	int tile_offset    = blockIdx.x * TILE_SIZE;
	int num_tiles      = (num_tuples + TILE_SIZE - 1) / TILE_SIZE;
	int num_tile_items = TILE_SIZE;

	if (blockIdx.x == num_tiles - 1) { num_tile_items = num_tuples - tile_offset; }

	InitFlags<BLOCK_THREADS, ITEMS_PER_THREAD>(selection_flags);

	BlockLoad<int, BLOCK_THREADS, ITEMS_PER_THREAD>(dim_key + tile_offset, items, num_tile_items);
	BlockLoad<int, BLOCK_THREADS, ITEMS_PER_THREAD>(dim_val + tile_offset, items2, num_tile_items);
	BlockBuildSelectivePHT_2<int, int, BLOCK_THREADS, ITEMS_PER_THREAD>(
	    items, items2, selection_flags, hash_table, num_slots, val_min, num_tile_items);
}

template <int BLOCK_THREADS, int ITEMS_PER_THREAD>
void runQuery(int*                         lo_orderdate,
              int*                         d_lo_orderdate_bw,
              int*                         d_lo_orderdate_base,
              int*                         d_lo_orderdate_offset,
              int*                         lo_partkey,
              int*                         lo_suppkey,
              int*                         lo_revenue,
              int                          lo_len,
              int*                         p_partkey,
              int*                         p_brand1,
              int*                         p_category,
              int                          p_len,
              int*                         d_datekey,
              int*                         d_year,
              int                          d_len,
              int*                         s_suppkey,
              int*                         s_region,
              int                          s_len,
              hipcub::CachingDeviceAllocator& g_allocator) {
	SETUP_TIMING();

	float time_query;

	hipEventRecord(start, 0);

	int *ht_d, *ht_p, *ht_s;
	int  d_val_len = 19981230 - 19920101 + 1;
	CubDebugExit(g_allocator.DeviceAllocate((void**)&ht_d, 2 * d_val_len * sizeof(int)));
	CubDebugExit(g_allocator.DeviceAllocate((void**)&ht_p, 2 * p_len * sizeof(int)));
	CubDebugExit(g_allocator.DeviceAllocate((void**)&ht_s, 2 * s_len * sizeof(int)));

	CubDebugExit(hipMemset(ht_d, 0, 2 * d_val_len * sizeof(int)));
	CubDebugExit(hipMemset(ht_p, 0, 2 * p_len * sizeof(int)));
	CubDebugExit(hipMemset(ht_s, 0, 2 * s_len * sizeof(int)));

	int tile_items = BLOCK_THREADS * ITEMS_PER_THREAD;

	build_hashtable_s<BLOCK_THREADS, ITEMS_PER_THREAD>
	    <<<(s_len + tile_items - 1) / tile_items, BLOCK_THREADS>>>(s_region, s_suppkey, s_len, ht_s, s_len);
	/*CHECK_ERROR();*/

	build_hashtable_p<BLOCK_THREADS, ITEMS_PER_THREAD>
	    <<<(p_len + tile_items - 1) / tile_items, BLOCK_THREADS>>>(p_category, p_partkey, p_brand1, p_len, ht_p, p_len);
	/*CHECK_ERROR();*/

	int d_val_min = 19920101;
	build_hashtable_d<BLOCK_THREADS, ITEMS_PER_THREAD><<<(d_len + tile_items - 1) / tile_items, BLOCK_THREADS>>>(
	    d_datekey, d_year, d_len, ht_d, d_val_len, d_val_min);
	/*CHECK_ERROR();*/

	int* res;
	int  res_size       = ((1998 - 1992 + 1) * (5 * 5 * 40));
	int  res_array_size = res_size * ITEMS_PER_THREAD;

	CubDebugExit(g_allocator.DeviceAllocate((void**)&res, res_array_size * sizeof(int)));
	CubDebugExit(hipMemset(res, 0, res_array_size * sizeof(int)));

	probe_v3<BLOCK_THREADS, ITEMS_PER_THREAD>
	    <<<(lo_len + tile_items - 1) / tile_items, BLOCK_THREADS>>>(lo_orderdate,
	                                                                d_lo_orderdate_bw,
	                                                                d_lo_orderdate_base,
	                                                                d_lo_orderdate_offset,
	                                                                lo_partkey,
	                                                                lo_suppkey,
	                                                                lo_revenue,
	                                                                lo_len,
	                                                                ht_s,
	                                                                s_len,
	                                                                ht_p,
	                                                                p_len,
	                                                                ht_d,
	                                                                d_val_len,
	                                                                res);

	int* h_res = new int[res_array_size];
	CubDebugExit(hipMemcpy(h_res, res, res_array_size * sizeof(int), hipMemcpyDeviceToHost));

	// int res_count = 0;
	// for (int i = 0; i < res_size; i++) {
	// 	if (h_res[4 * i] != 0) {
	// 		cout << h_res[4 * i] << " " << h_res[4 * i + 1] << " "
	// 		     << reinterpret_cast<unsigned long long*>(&h_res[4 * i + 2])[0] << endl;
	// 		res_count += 1;
	// 	}
	// }
	//
	// cout << "Res Count: " << res_count << endl;

	ssb::SSBQuery2ResultTable result_of_query;
	for (int i = 0; i < res_size; i++) {
		if (h_res[4 * i] != 0) {
			result_of_query.emplace_back(
			    h_res[4 * i], h_res[4 * i + 1], reinterpret_cast<unsigned long long*>(&h_res[4 * i + 2])[0]);
		}
	}

	ASSERT_EQ(result_of_query.size(), ssb::ssb_q21_10.reuslt.size());
	ASSERT_EQ(result_of_query, ssb::ssb_q21_10.reuslt);

	delete[] h_res;

	CLEANUP(res);
	CLEANUP(ht_d);
	CLEANUP(ht_p);
	CLEANUP(ht_s);
}

int main(int argc, char* argv[]) {
	auto hard_coded = query_mtd.ssb;

	int* h_lo_orderdate = loadColumn<int>("lo_orderdate", LO_LEN);
	int* h_lo_partkey   = loadColumn<int>("lo_partkey", LO_LEN);
	int* h_lo_suppkey   = loadColumn<int>("lo_suppkey", LO_LEN);
	int* h_lo_revenue   = loadColumn<int>("lo_revenue", LO_LEN);

	auto n_vec = hard_coded.n_vec;

	int* tmp = new int[n_vec * 1024];
	for (size_t i {0}; i < LO_LEN; ++i) {
		tmp[i] = h_lo_orderdate[i] - hard_coded.lo_orderdate_min;
	}

	const int* h_enc_lo_orderdate = new int[n_vec * 1024];
	const int* h_enc_lo_partkey   = new int[n_vec * 1024];
	const int* h_enc_lo_suppkey   = new int[n_vec * 1024];
	const int* h_enc_lo_revenue   = new int[n_vec * 1024];

	int* h_lo_orderdate_base   = new int[n_vec];
	int* h_lo_orderdate_bw     = new int[n_vec];
	int* h_lo_orderdate_offset = new int[n_vec];

	if (!is_sorted(h_lo_orderdate, LO_LEN)) {
		throw std::runtime_error("should be sorted!");
	}

	auto* orderdate_in = const_cast<int32_t*>(tmp);
	auto* partkey_in   = const_cast<int32_t*>(h_lo_partkey);
	auto* suppkey_in   = const_cast<int32_t*>(h_lo_suppkey);
	auto* revenue_in   = const_cast<int32_t*>(h_lo_revenue);

	auto* orderdate_out  = const_cast<int32_t*>(h_enc_lo_orderdate);
	auto* partkey_out   = const_cast<int32_t*>(h_enc_lo_partkey);
	auto* suppkey_out   = const_cast<int32_t*>(h_enc_lo_suppkey);
	auto* revenue_out   = const_cast<int32_t*>(h_enc_lo_revenue);

	constexpr int SF10_LAST_VECTOR_IDX = 58580;
	constexpr int LAST_VECTOR_SIZE     = 294;
	for (uint64_t vec_idx {0}; vec_idx < n_vec; vec_idx++) {
		if (vec_idx == SF10_LAST_VECTOR_IDX) { set_zero_after<1024>(orderdate_in, LAST_VECTOR_SIZE); }

		h_lo_orderdate_base[vec_idx] = find_base<1024>(orderdate_in);
		subtract_base<1024>(orderdate_in, h_lo_orderdate_base[vec_idx]);
		h_lo_orderdate_bw[vec_idx] = find_bw<1024>(orderdate_in);

		if (vec_idx + 1 < n_vec) {
			h_lo_orderdate_offset[vec_idx + 1] = h_lo_orderdate_offset[vec_idx] + (h_lo_orderdate_bw[vec_idx] * 32);
		}

		if (h_lo_orderdate_bw[vec_idx] > 16) {
			std::cout << h_lo_orderdate_bw[vec_idx] << " bigger than 16 is not possible in orderdate! \n";
			exit(-2);
		}

		generated::pack::fallback::scalar::pack(orderdate_in, orderdate_out, h_lo_orderdate_bw[vec_idx]);
		orderdate_in  = orderdate_in + 1024;
		orderdate_out = orderdate_out + (h_lo_orderdate_bw[vec_idx] * 32);

		generated::pack::fallback::scalar::pack(partkey_in, partkey_out, hard_coded.lo_partkey_bw);
		partkey_in  = partkey_in + 1024;
		partkey_out = partkey_out + (hard_coded.lo_partkey_bw * 32);

		generated::pack::fallback::scalar::pack(suppkey_in, suppkey_out, hard_coded.lo_chosen_suppkey_bw);
		suppkey_in  = suppkey_in + 1024;
		suppkey_out = suppkey_out + (hard_coded.lo_chosen_suppkey_bw * 32);

		generated::pack::fallback::scalar::pack(revenue_in, revenue_out, hard_coded.lo_revenue_bw);
		revenue_in  = revenue_in + 1024;
		revenue_out = revenue_out + (hard_coded.lo_revenue_bw * 32);
	}

	int* d_lo_orderdate = loadToGPU<int32_t>(h_enc_lo_orderdate, hard_coded.n_tup_line_order, g_allocator);
	int* d_lo_partkey   = loadToGPU<int32_t>(h_enc_lo_partkey, hard_coded.n_tup_line_order, g_allocator);
	int* d_lo_suppkey   = loadToGPU<int32_t>(h_enc_lo_suppkey, hard_coded.n_tup_line_order, g_allocator);
	int* d_lo_revenue   = loadToGPU<int32_t>(h_lo_revenue, hard_coded.n_tup_line_order, g_allocator);

	int* d_lo_orderdate_base   = loadToGPU<int32_t>(h_lo_orderdate_base, n_vec, g_allocator);
	int* d_lo_orderdate_bw     = loadToGPU<int32_t>(h_lo_orderdate_bw, n_vec, g_allocator);
	int* d_lo_orderdate_offset = loadToGPU<int32_t>(h_lo_orderdate_offset, n_vec, g_allocator);

	int* h_p_partkey  = loadColumn<int>("p_partkey", P_LEN);
	int* h_p_brand1   = loadColumn<int>("p_brand1", P_LEN);
	int* h_p_category = loadColumn<int>("p_category", P_LEN);

	int* h_d_datekey = loadColumn<int>("d_datekey", D_LEN);
	int* h_d_year    = loadColumn<int>("d_year", D_LEN);

	int* h_s_suppkey = loadColumn<int>("s_suppkey", S_LEN);
	int* h_s_region  = loadColumn<int>("s_region", S_LEN);

	int* d_d_datekey = loadToGPU<int>(h_d_datekey, D_LEN, g_allocator);
	int* d_d_year    = loadToGPU<int>(h_d_year, D_LEN, g_allocator);

	int* d_p_partkey  = loadToGPU<int>(h_p_partkey, P_LEN, g_allocator);
	int* d_p_brand1   = loadToGPU<int>(h_p_brand1, P_LEN, g_allocator);
	int* d_p_category = loadToGPU<int>(h_p_category, P_LEN, g_allocator);

	int* d_s_suppkey = loadToGPU<int>(h_s_suppkey, S_LEN, g_allocator);
	int* d_s_region  = loadToGPU<int>(h_s_region, S_LEN, g_allocator);

	runQuery<32, 8>(d_lo_orderdate,
	                d_lo_orderdate_bw,
	                d_lo_orderdate_base,
	                d_lo_orderdate_offset,
	                d_lo_partkey,
	                d_lo_suppkey,
	                d_lo_revenue,
	                LO_LEN,
	                d_p_partkey,
	                d_p_brand1,
	                d_p_category,
	                P_LEN,
	                d_d_datekey,
	                d_d_year,
	                D_LEN,
	                d_s_suppkey,
	                d_s_region,
	                S_LEN,
	                g_allocator);
}
