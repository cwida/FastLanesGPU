#include "hip/hip_runtime.h"
// Ensure printing of CUDA runtime errors to console
#define HIPCUB_STDERR

#include "cub/test/test_util.h"
#include "data/footer/ssb/ssb.hpp"
#include "fls_gen/unpack/hardcoded_16.cuh"
// #include "fls_gen/unpack/unpack_fused.cuh"
#include "crystal-opt/crystal.cuh"
#include "gpu_utils.h"
#include "ssb_utils.h"
#include <crystal_ssb_utils.h>
#include <fls_gen/pack/pack.hpp>
#include <iostream>
#include <stdio.h>

using namespace std;
using namespace fastlanes::gpu;
using namespace fastlanes;

constexpr uint32_t CONSTANT_1 = make_simd_const(10);
constexpr uint32_t CONSTANT_2 = make_simd_const(20);

template <int BLOCK_THREADS, int IPT>
__global__ void QueryKernel(const int*          enc_lo_orderdate,
                            const int*          enc_lo_discount,
                            const int*          enc_lo_quantity,
                            int*                lo_extendedprice,
                            ssb::SSB            query_mtd,
                            unsigned long long* revenue) {
	int TILE_SIZE = BLOCK_THREADS * IPT;
	// Load a segment of consecutive items that are blocked across threads
	uint16_t items[IPT];
	uint16_t selection_flags[IPT];

	int items2[IPT];

	long long sum = 0;

	int tile_offset    = blockIdx.x * TILE_SIZE;
	int num_tiles      = (query_mtd.n_tup_line_order + TILE_SIZE - 1) / TILE_SIZE;
	int num_tile_items = TILE_SIZE;
	if (blockIdx.x == num_tiles - 1) { num_tile_items = query_mtd.n_tup_line_order - tile_offset; }

	int orderdate_tile_offset = blockIdx.x * query_mtd.lo_orderdate_bw * 32;
	hardcoded::unpack_device(enc_lo_orderdate + orderdate_tile_offset, items, query_mtd.lo_orderdate_bw);

	for (size_t i {0}; i < 32; ++i) {
		items[i] = i;
		selection_flags[i] = 1;
	}
	// BlockPredGT_int_16_2<BLOCK_THREADS, IPT>(
	//     //
	//     reinterpret_cast<uint32_t(&)[IPT]>(items),
	//     CONSTANT_1,
	//     reinterpret_cast<uint32_t(&)[IPT]>(selection_flags),
	//     num_tile_items);
	BlockPredAndLTX<BLOCK_THREADS, IPT>(
	    //
	    reinterpret_cast<uint32_t(&)[IPT]>(items),
	    CONSTANT_2,
	    reinterpret_cast<uint32_t(&)[IPT]>(selection_flags),
	    num_tile_items);

	for (size_t i {0}; i < 32; ++i) {
	printf("%d\n", selection_flags[i]);
	}



	int quantity_tile_offset = blockIdx.x * query_mtd.lo_quantity_bw * 32;
	hardcoded::unpack_device(enc_lo_quantity + quantity_tile_offset, items, query_mtd.lo_quantity_bw);
	BlockPredAndLT<uint16_t, uint16_t, BLOCK_THREADS, IPT>(items, 25, selection_flags, num_tile_items);

	int discount_tile_offset = blockIdx.x * query_mtd.lo_discount_bw * 32;
	hardcoded::unpack_device(enc_lo_discount + discount_tile_offset, items, query_mtd.lo_discount_bw);
	BlockPredAndGTE<uint16_t, uint16_t, BLOCK_THREADS, IPT>(items, 1, selection_flags, num_tile_items);
	BlockPredAndLTE<uint16_t, uint16_t, BLOCK_THREADS, IPT>(items, 3, selection_flags, num_tile_items);

	BlockPredLoad<int, uint16_t, BLOCK_THREADS, IPT>(
	    lo_extendedprice + tile_offset, items2, num_tile_items, selection_flags);

#pragma unroll
	for (int ITEM = 0; ITEM < IPT; ++ITEM) {
		if ((threadIdx.x + (BLOCK_THREADS * ITEM) < num_tile_items))
			if (selection_flags[ITEM]) sum += items[ITEM] * items2[ITEM];
	}

	__syncthreads();

	static __shared__ long long buffer[32];
	unsigned long long          aggregate = BlockSum<long long, BLOCK_THREADS, IPT>(sum, (long long*)buffer);
	__syncthreads();

	if (threadIdx.x == 0) { atomicAdd(revenue, aggregate); }
}

template <int BLOCK_THREADS, int ITEMS_PER_THREAD>
float query(int*                         lo_orderdate,
            int*                         lo_discount,
            int*                         lo_quantity,
            int*                         lo_extendedprice,
            ssb::SsbQueryMtd             query_mtd,
            hipcub::CachingDeviceAllocator& g_allocator) {
	SETUP_TIMING();

	float                                     time_query;
	chrono::high_resolution_clock::time_point st, finish;
	st = chrono::high_resolution_clock::now();

	hipEventRecord(start, 0);

	unsigned long long* d_sum = NULL;
	CubDebugExit(g_allocator.DeviceAllocate((void**)&d_sum, sizeof(long long)));

	hipMemset(d_sum, 0, sizeof(long long));

	// Run
	int tile_items = BLOCK_THREADS * ITEMS_PER_THREAD;
	int num_blocks = (query_mtd.ssb.n_tup_line_order + tile_items - 1) / tile_items;
	QueryKernel<BLOCK_THREADS, ITEMS_PER_THREAD>
	    <<<1, 1>>>(lo_orderdate, lo_discount, lo_quantity, lo_extendedprice, query_mtd.ssb, d_sum);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time_query, start, stop);

	unsigned long long revenue;
	CubDebugExit(hipMemcpy(&revenue, d_sum, sizeof(long long), hipMemcpyDeviceToHost));

	finish                             = chrono::high_resolution_clock::now();
	std::chrono::duration<double> diff = finish - st;

	double total_time_taken {diff.count() * 1000};
	FLS_SHOW(total_time_taken)

	/*Check the result*/
	FLS_SHOW(revenue)
	if (revenue != query_mtd.result) { throw std::runtime_error("RESULT INCOREECT!"); }
	FLS_SUCCESS(query_mtd.ssb.name)

	CLEANUP(d_sum);

	return time_query;
}

int main() {
	int  num_trials  = 3;
	auto queries_mtd = {
	    //
	    ssb::ssb_q11_10,
	    //
	};
	for (const auto query_mtd : queries_mtd) {
		auto hard_coded         = query_mtd.ssb;
		int* h_lo_orderdate     = loadColumn<int>("lo_orderdate", LO_LEN);
		int* h_lo_discount      = loadColumn<int>("lo_discount", LO_LEN);
		int* h_lo_quantity      = loadColumn<int>("lo_quantity", LO_LEN);
		int* h_lo_extendedprice = loadColumn<int>("lo_extendedprice", LO_LEN);

		auto n_vec = hard_coded.n_vec;

		int* tmp = new int[n_vec * 1024];
		for (size_t i {0}; i < LO_LEN; ++i) {
			tmp[i] = h_lo_orderdate[i] - hard_coded.lo_orderdate_min;
		}

		const int* h_enc_lo_orderdate = new int[n_vec * 1024];
		const int* h_enc_lo_discount  = new int[n_vec * 1024];
		const int* h_enc_lo_quantity  = new int[n_vec * 1024];

		auto* orderdate_in = const_cast<const int32_t*>(tmp);
		auto* discount_in  = const_cast<int32_t*>(h_lo_discount);
		auto* quantity_in  = const_cast<int32_t*>(h_lo_quantity);

		auto* orderdate_out = const_cast<int32_t*>(h_enc_lo_orderdate);
		auto* discount_out  = const_cast<int32_t*>(h_enc_lo_discount);
		auto* quantity_out  = const_cast<int32_t*>(h_enc_lo_quantity);

		for (uint64_t vec_idx {0}; vec_idx < n_vec; vec_idx++) {
			generated::pack::fallback::scalar::pack(orderdate_in, orderdate_out, hard_coded.lo_orderdate_bw);
			orderdate_in  = orderdate_in + 1024;
			orderdate_out = orderdate_out + (hard_coded.lo_orderdate_bw * 32);

			generated::pack::fallback::scalar::pack(discount_in, discount_out, hard_coded.lo_discount_bw);
			discount_in  = discount_in + 1024;
			discount_out = discount_out + (hard_coded.lo_discount_bw * 32);

			generated::pack::fallback::scalar::pack(quantity_in, quantity_out, hard_coded.lo_quantity_bw);
			quantity_in  = quantity_in + 1024;
			quantity_out = quantity_out + (hard_coded.lo_quantity_bw * 32);
		}

		FLS_LOG("LOADED DATA")

		int* d_lo_orderdate     = loadToGPU<int32_t>(h_enc_lo_orderdate, hard_coded.n_tup_line_order, g_allocator);
		int* d_lo_discount      = loadToGPU<int32_t>(h_enc_lo_discount, hard_coded.n_tup_line_order, g_allocator);
		int* d_lo_quantity      = loadToGPU<int32_t>(h_enc_lo_quantity, hard_coded.n_tup_line_order, g_allocator);
		int* d_lo_extendedprice = loadToGPU<int32_t>(h_lo_extendedprice, hard_coded.n_tup_line_order, g_allocator);

		FLS_LOG("LOADED DATA TO GPU")

		for (int n = 0; n < num_trials; n++) {
			auto t =
			    query<1, 32>(d_lo_orderdate, d_lo_discount, d_lo_quantity, d_lo_extendedprice, query_mtd, g_allocator);
			FLS_RESULT(t)
		}
	}
	return 0;
}